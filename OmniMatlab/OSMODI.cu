#include "hip/hip_runtime.h"
/*
 * © 2024. Triad National Security, LLC. All rights reserved.
 * This program was produced under U.S. Government contract
 * 89233218CNA000001 for Los Alamos National Laboratory (LANL), which is
 * operated by Triad National Security, LLC for the U.S. Department of
 * Energy/National Nuclear Security Administration. All rights in the
 * program are reserved by Triad National Security, LLC, and the U.S.
 * Department of Energy/National Nuclear Security Administration. The
 * Government is granted for itself and others acting on its behalf a
 * nonexclusive, paid-up, irrevocable worldwide license in this material
 * to reproduce, prepare. derivative works, distribute copies to the
 * public, perform publicly and display publicly, and to permit
 * others to do so.
 *
 * This program is free software: you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your
 * option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program. If not, see <https://www.gnu.org/licenses/>.
 */

//======================================================================
//======================================================================
//====================OMNIDIRECTIONAL CUDA MATRIX SOLVER================
//======================================================================
//======================================================================
//Developed by Fernando Zigunov and John Charonko (2023) - Extreme Fluids Group - Los Alamos National Laboratory
//V01 - This is the ONE-SHOT version of the omnidirectional matrix solver. (i.e., one solution of the CG solver solves for the correct pressure field.)
//R02 - Added preconditioner to CG solver
//====This is a Matlab implementation using mexcuda.====
//The variables are transferred directly from Matlab memory to here.
//Compile with "mexcuda OSMODI.cu"
//
//Call this function as:
//[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz); %Basic form, source term Sx, Sy, Sz are either 2D matrices or 3D matrices. 
//                     We use **ND grid** format here. Uses default options, and delta=1.
//                     If the fields are 2D, please provide Sz anyways, it can be zeros(size(Sx)).
//[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, delta); %Also provides a grid spacing delta which is the same in all directions.
//[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, [dx dy dz]); %Provides a grid spacing that is different for x, y, z but is still constant for each direction.
//[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, delta, options); %Also provides options as a struct. See below.

//options.SolverToleranceRel (default is 1e-4) %Relative error allowed for the CG solver
//options.SolverToleranceAbs (default is 1e-4) %Relative error allowed for the CG solver
//options.SolverDevice (default is 'GPU') %Choose between 'CPU' and 'GPU'.
//options.Verbose (default is '0') %To make the code print the output (Verbose=1 prints).


#pragma once

#include "mex.h"
#include <iostream>
#include <regex>
#include <fstream> 
#include <string>
#include <vector>
#include <iterator>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <algorithm>
#include <sstream>
#include <iomanip>
#include <omp.h>
#include <random>
#include <signal.h>

using namespace std;

#define PI 3.141592653589793
#define _X 0
#define _Y 1
#define _Z 2

#define ZEROS 0
#define ONES 1
#define RANDOMS 2

#define CPU 0
#define GPU 1

#define BLOCKDIM_VEC 512

clock_t tic; clock_t toc; double timeTask;
clock_t tic2; clock_t toc2; double timeTask2;

typedef float varfloat; //Change this to "float" for single precision or "double" for double precision. Found that "float" is ~5-7% faster (not really all that much)

// ====================Structure Definitions================ 
#pragma region

template <typename varfloat>
struct varfloat3 {
    varfloat x;
    varfloat y;
    varfloat z;
};

template <typename varfloat>
struct SolverParameters {
    bool Verbose = false; //Whether to send messages to Matlab console
    int SolverDevice = GPU; //CPU or GPU
    varfloat solverToleranceRel = 1e-4; //Error allowed for the solver
    varfloat solverToleranceAbs = 1e-4; //Error allowed for the solver
    int3 BoxGridPoints = { 100, 100, 1 }; //Number of grid points in the box
    varfloat3<varfloat> GridDelta; //Delta value for derivative approximation
    long long totalBoxElements; //Tracks the size of the box
} ; // Structure to define the parameters of the solver; initializes to default values

template <typename varfloat>
struct BoxContents {
    varfloat* SourceFn_Field_X; //Stores the Source Function here
    varfloat* SourceFn_Field_Y; //Stores the Source Function here
    varfloat* SourceFn_Field_Z; //Stores the Source Function here
    int3 BoxGridSize = { 100, 100, 1 }; //Number of grid points in the box
    varfloat3<varfloat> GridDelta; //Delta value for derivative approximation
    long long totalBoxElements; //Tracks the size of the box
} ; // Structure to hold the contents of the 3D box 

struct Progress { //Struct to store the progress of the CG solver
    int Iteration;
    float Residual;
    double TimeSeconds;
};

vector<Progress> CGS_Progress;

#pragma endregion



// ====================Helper Functions================ 
#pragma region

bool iequals(const string& a, const string& b)
{
    //Adapted from answers in
    //https://stackoverflow.com/questions/11635/case-insensitive-string-comparison-in-c
    return std::equal(a.begin(), a.end(),
        b.begin(), b.end(),
        [](char a, char b) {
            return tolower(a) == tolower(b);
        });
}

template <typename varfloat>
__host__ __device__ inline int isnan2(varfloat x)
{
    //Apparently uses less registers than the original isnan, See:
    //https://stackoverflow.com/questions/33922103/is-isnan2-as-fast-as-testing-equality
    return x != x;
}

void ClockTic() {
    //Starts the clock
    tic = clock();
}

double ClockToc() {
    //Returns the current clock time  in seconds
    toc = clock() - tic;
    timeTask = ((double)toc) / CLOCKS_PER_SEC; // in seconds
    return timeTask;
}

template <typename varfloat>
void FillBox(varfloat* boxToFill, int BoxContents, SolverParameters<varfloat> SP) {
    //Fills the 3D box with one of the objects as dictated by BoxContents
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> distribution(-1.0f, 1.0f);

    for (int zz = 0; zz < SP.BoxGridPoints.z; zz++) {
        for (int yy = 0; yy < SP.BoxGridPoints.y; yy++) {
            for (int xx = 0; xx < SP.BoxGridPoints.x; xx++) {
                long long idxBox = (long long)xx + (long long)SP.BoxGridPoints.x * ((long long)yy + (long long)SP.BoxGridPoints.y * ((long long)zz));

                if (BoxContents == ZEROS) {
                    boxToFill[idxBox] = 0.0f;
                }
                else if (BoxContents == ONES) {
                    boxToFill[idxBox] = 1.0f;
                }
                else if (BoxContents == RANDOMS) {
                    boxToFill[idxBox] = distribution(gen);
                }
                else {
                    boxToFill[idxBox] = 0.0f;
                }
            }
        }
    }
}

bool InitializeGPU(SolverParameters<varfloat> SP) {
    //This function initializes the GPU code by first enumerating the devices available and choosing the highest one for these computations.
    //Returns true if initialization was successful.

    if (SP.Verbose){ mexPrintf("Detecting GPUs...\n");}
    int NumberOfGPUs;
    hipGetDeviceCount(&NumberOfGPUs);

    if (NumberOfGPUs == 0) {
        return false; //No GPU available
        mexPrintf("\n ***Warning! No GPU found in the system!***\n");
    }

    size_t HighestGlobalMem = 0;
    string LargestDevName;
    int LargestDevID = -1;
    for (int i = 0; i < NumberOfGPUs; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        size_t ThisDeviceMem = prop.totalGlobalMem;
        if (HighestGlobalMem < ThisDeviceMem) {
            HighestGlobalMem = ThisDeviceMem;
            LargestDevID = i;
            LargestDevName = prop.name;
        }

        if (SP.Verbose){ mexPrintf("Device Number: %d\n", i);}
        if (SP.Verbose){ mexPrintf("  Device name: %s\n", prop.name);}
        if (SP.Verbose){ mexPrintf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate / 1.0e3);}
        if (SP.Verbose){ mexPrintf("  Device Global Memory (GB): %f\n\n", ThisDeviceMem / 1.0e9);}
    }

    if (LargestDevID >= 0) {
        if (SP.Verbose){ mexPrintf("\nUsing Device: %s\n", LargestDevName.c_str());}
        hipSetDevice(LargestDevID);
        return true;
    }
    else{
        printf("\n***Warning! Found GPU device but for some reason code failed!***\n");
        return false;
    }

}

#pragma endregion

// ====================GPU CUDA Function Kernels================
#pragma region

template <typename varfloat>
__global__ void GPU_FillNan(varfloat* PressureField, SolverParameters<varfloat>* SP) {
    //Fills the box with nan values in case the CG diverges
    long long xx = blockIdx.x * blockDim.x + threadIdx.x;
    long long yy = blockIdx.y * blockDim.y + threadIdx.y;
    long long zz = blockIdx.z * blockDim.z + threadIdx.z;

    if ((xx >= SP->BoxGridPoints.x) || (yy >= SP->BoxGridPoints.y) || (zz >= SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }

    long long GridX = (long long)SP->BoxGridPoints.x;
    long long GridY = (long long)SP->BoxGridPoints.y;
    long long idxCenter = xx + GridX * (yy + GridY * zz);
    PressureField[idxCenter] = NAN;
}

template <typename varfloat>
__global__ void printScalar_GPU(varfloat* a) {
    //For debugging purposes
    printf("%f\n", *a);
}

template <typename varfloat>
__global__ void printVector1_GPU(varfloat* v) {
    //For debugging purposes
    printf("%e; ", v[0]);
}

template <typename varfloat>
__global__ void printVector2_GPU(varfloat* v) {
    //For debugging purposes
    printf("%e\n", v[49]);
}

template <typename varfloat>
__global__ void addVectors_GPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    long long idxCenter = (long long)blockIdx.x * (long long)blockDim.x + (long long)threadIdx.x;

    if (idxCenter >= (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }
    out[idxCenter] = a[idxCenter] + b[idxCenter];
}

template <typename varfloat>
__global__ void subtractVectors_GPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    long long idxCenter = (long long)blockIdx.x * (long long)blockDim.x + (long long)threadIdx.x;

    if (idxCenter >= (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }
    out[idxCenter] = a[idxCenter] - b[idxCenter];
}

template <typename varfloat>
__global__ void divide(varfloat* num, varfloat* den, varfloat* out) {
    unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (index_x == 0) {
        *out = *num / *den;
    }
}

template <typename varfloat>
__global__ void vectorDot_GPU_Slow(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    //naive slow implementation
    unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    varfloat tmp = 0.0;
    if (index_x == 0) {
        for (long long i = 0; i < (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z); i++) {
            if (!isnan2(a[i]) && !isnan2(b[i])) {
                tmp += a[i] * b[i];
                //printf("i=%lld; a=%f, b=%f, tmp=%f, \n", i, a[i], b[i],tmp);
            }
        }
        *out = tmp;
    }
}

template <typename varfloat>
__global__ void scalarVectorMult_GPU(varfloat* scalar, varfloat* a, varfloat* out, SolverParameters<varfloat>* SP) {
    unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (index_x < (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z)) {
        out[index_x] = a[index_x] * *scalar;
    }
}

template <typename varfloat>
__global__ void vectorDot_GPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    __shared__ varfloat shared_tmp[BLOCKDIM_VEC];

    long long index = (long long)blockIdx.x * (long long)blockDim.x + (long long)threadIdx.x;
    long long maxIdx = (long long)SP->BoxGridPoints.x * (long long)SP->BoxGridPoints.y * (long long)SP->BoxGridPoints.z;

    if (index == 0) {
        *out = 0.0; //We need to reset the output variable when we start
    }

    if (index < maxIdx) {
        if (!isnan2(a[index]) || !isnan2(b[index])) {
            shared_tmp[threadIdx.x] = a[index] * b[index]; //Dont norm nans
        }
        else {
            shared_tmp[threadIdx.x] = 0.0;//nans become zeros
        }
    }
    else {
        shared_tmp[threadIdx.x] = 0.0;
    }

    __syncthreads();

    // reduction within block
    unsigned int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            shared_tmp[threadIdx.x] += shared_tmp[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    // atomic add the partial reduction in out
    if (threadIdx.x == 0) {
        atomicAdd(out, shared_tmp[0]);
    }
}

template <typename varfloat>
__global__ void MatrixMul_Omnidirectional_GPU(varfloat* Result, varfloat* PressureField, varfloat* RHS, SolverParameters<varfloat>* SP) {
    //This is the bit of code that performs the matrix multiplication Result=A*x (where A is the weight matrix and x is the PressureField)
    //The RHS of the equation is also provided so we can find the points where we have NAN's
    long long xx = blockIdx.x * blockDim.x + threadIdx.x;
    long long yy = blockIdx.y * blockDim.y + threadIdx.y;
    long long zz = blockIdx.z * blockDim.z + threadIdx.z;

    if ((xx >= SP->BoxGridPoints.x) || (yy >= SP->BoxGridPoints.y) || (zz >= SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }

    if (SP->BoxGridPoints.z == 1) {
        //2D Case
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;

        long long idxCenter = xx + GridX * (yy + GridY * zz);

        //dx and dy for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;

        if ((RHS[idxCenter] != RHS[idxCenter])) {
            //The RHS here is a nan, so simply makes the result at this point a nan as well
            Result[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_xm = idxCenter - 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_ym = idxCenter - GridX;

            varfloat bxp = ((xx + 1) >= GridX) || (RHS[idx_xp] != RHS[idx_xp]); // isnans exposed as inequalities to reduce the number of registers required (from 112 to 56) [i.e. isnan(X) is the same as X!=X]
            varfloat byp = ((yy + 1) >= GridY) || (RHS[idx_yp] != RHS[idx_yp]);
            varfloat bxm = ((xx - 1) < 0) || (RHS[idx_xm] != RHS[idx_xm]);
            varfloat bym = ((yy - 1) < 0) || (RHS[idx_ym] != RHS[idx_ym]);

            varfloat rhs_cx = GridDY / (2.0 * (GridDX + GridDY));
            varfloat rhs_cy = GridDX / (2.0 * (GridDX + GridDY));

            //Adds the pressure values to right-hand side for this cell 
            varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym); //Weight for the center coefficient
            varfloat w_in_1 = 1.0 - w_in;

            varfloat R = PressureField[idxCenter];
            R -= bxp ? 0.0 : rhs_cx * PressureField[idx_xp] / w_in_1; //done this way to prevent access outside allocated memory 
            R -= bxm ? 0.0 : rhs_cx * PressureField[idx_xm] / w_in_1;
            R -= byp ? 0.0 : rhs_cy * PressureField[idx_yp] / w_in_1;
            R -= bym ? 0.0 : rhs_cy * PressureField[idx_ym] / w_in_1;
            Result[idxCenter] = R;
        }
    }
    else {
        //3D Case
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long GridZ = (long long)SP->BoxGridPoints.z;

        long long idxCenter = xx + GridX * (yy + GridY * zz);

        //dx and dy and dz for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;
        varfloat GridDZ = SP->GridDelta.z;

        if (RHS[idxCenter] != RHS[idxCenter]) {
            //The RHS here is a nan, so simply makes the result at this point a nan as well
            Result[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_xm = idxCenter - 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_ym = idxCenter - GridX;
            long long idx_zp = idxCenter + GridX * GridY;
            long long idx_zm = idxCenter - GridX * GridY;

            varfloat bxp = ((xx + 1) >= GridX) || (RHS[idx_xp] != RHS[idx_xp]); // isnans exposed as inequalities to reduce the number of registers required (from 112 to 80) [i.e. isnan(X) is the same as X!=X]
            varfloat byp = ((yy + 1) >= GridY) || (RHS[idx_yp] != RHS[idx_yp]);
            varfloat bzp = ((zz + 1) >= GridZ) || (RHS[idx_zp] != RHS[idx_zp]);
            varfloat bxm = ((xx - 1) < 0) || (RHS[idx_xm] != RHS[idx_xm]);
            varfloat bym = ((yy - 1) < 0) || (RHS[idx_ym] != RHS[idx_ym]);
            varfloat bzm = ((zz - 1) < 0) || (RHS[idx_zm] != RHS[idx_zm]);

            //Computes the weights for the [n] coefficients
            varfloat rhs_den = 2.0 * (GridDX * GridDY + GridDX * GridDZ + GridDY * GridDZ);
            varfloat rhs_cx = (GridDY * GridDZ) / rhs_den;
            varfloat rhs_cy = (GridDX * GridDZ) / rhs_den;
            varfloat rhs_cz = (GridDX * GridDY) / rhs_den;

            //Adds the pressure values to right-hand side for this cell 
            varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym) + rhs_cz * (bzp + bzm); //Weight for the center coefficient
            varfloat w_in_1 = 1.0 - w_in;

            varfloat R = PressureField[idxCenter];
            R -= bxp ? 0 : rhs_cx * PressureField[idx_xp] / w_in_1; //done this way to prevent access outside allocated memory 
            R -= bxm ? 0 : rhs_cx * PressureField[idx_xm] / w_in_1;
            R -= byp ? 0 : rhs_cy * PressureField[idx_yp] / w_in_1;
            R -= bym ? 0 : rhs_cy * PressureField[idx_ym] / w_in_1;
            R -= bzp ? 0 : rhs_cz * PressureField[idx_zp] / w_in_1;
            R -= bzm ? 0 : rhs_cz * PressureField[idx_zm] / w_in_1;
            Result[idxCenter] = R;
        }

    }

}

template <typename varfloat>
__global__ void UpdateRHS_Vector_GPU(varfloat* PressureField, varfloat* RHS, varfloat* SourceX, varfloat* SourceY, varfloat* SourceZ, SolverParameters<varfloat>* SP) {
    //Computes the right-hand side vector based on the values of the pressures for all cells, considering boundaries, etc.
    long long xx = blockIdx.x * blockDim.x + threadIdx.x;
    long long yy = blockIdx.y * blockDim.y + threadIdx.y;
    long long zz = blockIdx.z * blockDim.z + threadIdx.z;

    if ((xx >= SP->BoxGridPoints.x) || (yy >= SP->BoxGridPoints.y) || (zz >= SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }

    if (SP->BoxGridPoints.z == 1) {
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long idxCenter = xx + GridX * (yy + GridY * zz);

        //dx and dy for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;

        if (SourceX[idxCenter] != SourceX[idxCenter]) {
            //The source value here is a nan, so simply makes the RHS at this point a nan as well
            RHS[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_xm = idxCenter - 1;
            long long idx_ym = idxCenter - GridX;

            //Computes the boolean values for each index
            varfloat bxp = ((xx + 1) >= GridX) || (SourceX[idx_xp] != SourceX[idx_xp]);
            varfloat byp = ((yy + 1) >= GridY) || (SourceX[idx_yp] != SourceX[idx_yp]);
            varfloat bxm = ((xx - 1) < 0) || (SourceX[idx_xm] != SourceX[idx_xm]);
            varfloat bym = ((yy - 1) < 0) || (SourceX[idx_ym] != SourceX[idx_ym]);

            //Computes the weights for the [n] coefficients
            //varfloat wxmax = gc->wxx + 2 * gc->wxy; varfloat wymax = gc->wyy + 2 * gc->wxy; //Weights for out-of-bounds conditions
            varfloat rhs_cx = GridDY / (2.0 * (GridDX + GridDY));
            varfloat rhs_cy = GridDX / (2.0 * (GridDX + GridDY));

            varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym); //Weight for the center coefficient
            varfloat w_in_1 = 1.0 - w_in;

            //Adds the pressure values to right-hand side for this cell
            varfloat R = 0.0;
            R += bxp ? 0.0 : (- rhs_cx * (SourceX[idx_xp] + SourceX[idxCenter]) * (GridDX / 2.0)) / w_in_1;
            R += bxm ? 0.0 : (rhs_cx * (SourceX[idx_xm] + SourceX[idxCenter]) * (GridDX / 2.0)) / w_in_1;
            R += byp ? 0.0 : (- rhs_cy * (SourceY[idx_yp] + SourceY[idxCenter]) * (GridDY / 2.0)) / w_in_1;
            R += bym ? 0.0 : (rhs_cy * (SourceY[idx_ym] + SourceY[idxCenter]) * (GridDY / 2.0)) / w_in_1;
            RHS[idxCenter] = R;
        }
    }
    else {
        //3D case
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long GridZ = (long long)SP->BoxGridPoints.z;
        long long idxCenter = xx + GridX * (yy + GridY * zz);

        //dx and dy and dz for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;
        varfloat GridDZ = SP->GridDelta.z;

        if (SourceX[idxCenter] != SourceX[idxCenter]) {
            //The source value here is a nan, so simply makes the RHS at this point a nan as well
            RHS[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_xm = idxCenter - 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_ym = idxCenter - GridX;
            long long idx_zp = idxCenter + GridX * GridY;
            long long idx_zm = idxCenter - GridX * GridY;

            //Computes the boolean values for each index
            varfloat bxp = ((xx + 1) >= GridX) || (SourceX[idx_xp] != SourceX[idx_xp]);
            varfloat bxm = ((xx - 1) < 0) || (SourceX[idx_xm] != SourceX[idx_xm]);
            varfloat byp = ((yy + 1) >= GridY) || (SourceX[idx_yp] != SourceX[idx_yp]);
            varfloat bym = ((yy - 1) < 0) || (SourceX[idx_ym] != SourceX[idx_ym]);
            varfloat bzp = ((zz + 1) >= GridZ) || (SourceX[idx_zp] != SourceX[idx_zp]);
            varfloat bzm = ((zz - 1) < 0) || (SourceX[idx_zm] != SourceX[idx_zm]);

            //Computes the weights for the [n] coefficients
            varfloat rhs_den = 2.0 * (GridDX * GridDY + GridDX * GridDZ + GridDY * GridDZ);
            varfloat rhs_cx = (GridDY * GridDZ) / rhs_den;
            varfloat rhs_cy = (GridDX * GridDZ) / rhs_den;
            varfloat rhs_cz = (GridDX * GridDY) / rhs_den;
            
            varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym) + rhs_cz * (bzp + bzm); //Weight for the center coefficient
            varfloat w_in_1 = 1.0 - w_in;

            //Adds the pressure values to right-hand side for this cell   
            varfloat R = 0.0;
            R += bxp ? 0.0 : (- rhs_cx * (SourceX[idx_xp] + SourceX[idxCenter]) * GridDX / 2) / w_in_1;
            R += bxm ? 0.0 : (rhs_cx * (SourceX[idx_xm] + SourceX[idxCenter]) * GridDX / 2) / w_in_1;
            R += byp ? 0.0 : (- rhs_cy * (SourceY[idx_yp] + SourceY[idxCenter]) * GridDY / 2) / w_in_1;
            R += bym ? 0.0 : (rhs_cy * (SourceY[idx_ym] + SourceY[idxCenter]) * GridDY / 2) / w_in_1;
            R += bzp ? 0.0 : (- rhs_cz * (SourceZ[idx_zp] + SourceZ[idxCenter]) * GridDZ / 2) / w_in_1;
            R += bzm ? 0.0 : (rhs_cz * (SourceZ[idx_zm] + SourceZ[idxCenter]) * GridDZ / 2) / w_in_1;
            RHS[idxCenter] = R;
        }
    }
}

template <typename varfloat>
void ConjugateGradientSolver_GPU(varfloat* PressureField, varfloat* RHS, SolverParameters<varfloat> SolverConfig, BoxContents<varfloat> VTK_Contents) {
    // Allocate GPU memory for source field and pressure field
    hipFree(0); //Initializes GPU context    
    
    //Creates concurrent streams so processing can occur in parallel
    const int nStreams = 4;
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; ++i) {
        hipStreamCreate(&stream[i]);
    }

    //Memory allocation
    long long boxArraySize = sizeof(varfloat) * VTK_Contents.totalBoxElements;
    varfloat* d_PressureField; varfloat* d_RHS; 
    varfloat* d_rk; varfloat* d_rkp1; varfloat* d_pk; varfloat* d_temp;
    hipMalloc(&d_PressureField, boxArraySize); hipMalloc(&d_RHS, boxArraySize);
    hipMalloc(&d_rk, boxArraySize); hipMalloc(&d_rkp1, boxArraySize);
    hipMalloc(&d_pk, boxArraySize); hipMalloc(&d_temp, boxArraySize);

    hipMemcpyAsync(d_PressureField, PressureField, boxArraySize, hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_RHS, RHS, boxArraySize, hipMemcpyHostToDevice, stream[1]);

    //Allocate GPU memory for the source terms
    varfloat* d_SourceX; varfloat* d_SourceY; varfloat* d_SourceZ;
    hipMalloc(&d_SourceX, boxArraySize); hipMalloc(&d_SourceY, boxArraySize); hipMalloc(&d_SourceZ, boxArraySize);
    hipMemcpyAsync(d_SourceX, VTK_Contents.SourceFn_Field_X, boxArraySize, hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_SourceY, VTK_Contents.SourceFn_Field_Y, boxArraySize, hipMemcpyHostToDevice, stream[1]);
    hipMemcpyAsync(d_SourceZ, VTK_Contents.SourceFn_Field_Z, boxArraySize, hipMemcpyHostToDevice, stream[2]);

    //Allocates auxiliary variables
    SolverParameters<varfloat>* d_SolverConfig;
    hipMalloc(&d_SolverConfig, sizeof(SolverParameters<varfloat>));
    hipMemcpyAsync(d_SolverConfig, &SolverConfig, sizeof(SolverParameters<varfloat>), hipMemcpyHostToDevice, stream[1]);

    //Allocates scalars
    varfloat* d_beta; varfloat* d_alpha; varfloat* d_r_norm; varfloat* d_r_norm_old; varfloat* d_temp_scal; 
    hipMalloc((void**)&d_beta, sizeof(varfloat)); hipMalloc((void**)&d_alpha, sizeof(varfloat)); hipMalloc((void**)&d_r_norm, sizeof(varfloat));
    hipMalloc((void**)&d_r_norm_old, sizeof(varfloat)); hipMalloc((void**)&d_temp_scal, sizeof(varfloat));

    dim3 threadsPerBlock3D; dim3 numBlocks3D; //3D for the matrix kernel
    if (VTK_Contents.BoxGridSize.z == 1) {
        threadsPerBlock3D = dim3(16, 16, 1);
        numBlocks3D = dim3(ceil(VTK_Contents.BoxGridSize.x / (varfloat)threadsPerBlock3D.x), ceil(VTK_Contents.BoxGridSize.y / (varfloat)threadsPerBlock3D.y), 1);
    }
    else {
        threadsPerBlock3D = dim3(4, 8, 8);
        numBlocks3D = dim3(ceil(VTK_Contents.BoxGridSize.x / (varfloat)threadsPerBlock3D.x), ceil(VTK_Contents.BoxGridSize.y / (varfloat)threadsPerBlock3D.y), ceil(VTK_Contents.BoxGridSize.z / (varfloat)threadsPerBlock3D.z));
    }
    dim3 threadsPerBlock1D = dim3(BLOCKDIM_VEC, 1, 1); //1D for the vector kernels
    dim3 numBlocks1D = dim3(ceil(VTK_Contents.totalBoxElements / (varfloat)threadsPerBlock1D.x), 1, 1);

    ClockTic();
    double IdleClock = 0.0; // to keep track of the time so every 10 seconds it still prints something
    //=====Updates RHS terms=====
    hipDeviceSynchronize();
    UpdateRHS_Vector_GPU << <numBlocks3D, threadsPerBlock3D, 0, stream[0] >> > (d_PressureField, d_RHS, d_SourceX, d_SourceY, d_SourceZ, d_SolverConfig); //b

    //=====Starts CG solver computations=====
    hipDeviceSynchronize();
    MatrixMul_Omnidirectional_GPU << <numBlocks3D, threadsPerBlock3D, 0, stream[0] >> > (d_temp, d_PressureField, d_RHS, d_SolverConfig); //temp=A*x_0
    hipDeviceSynchronize();
    subtractVectors_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_RHS, d_temp, d_rk, d_SolverConfig); //r_0=b-A*x_0

    hipDeviceSynchronize();
    hipMemcpyAsync(d_pk, d_rk, boxArraySize, hipMemcpyDeviceToDevice, stream[0]); //p_0=r_0
    vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_rk, d_rk, d_r_norm_old, d_SolverConfig); //r_k dot r_kvarfloat r_norm; 
    hipDeviceSynchronize();

    varfloat r_norm_init;
    Progress P_cgs;
    hipMemcpy(&r_norm_init, d_r_norm_old, sizeof(varfloat), hipMemcpyDeviceToHost); // initial residual norm
    hipDeviceSynchronize();
    r_norm_init = sqrt(r_norm_init);

    if (SolverConfig.Verbose){ mexPrintf("Initial Residual Norm=%f\n", r_norm_init);}
    CGS_Progress.clear();
    P_cgs.Iteration = 0; P_cgs.Residual = 1.0f;  P_cgs.TimeSeconds = ClockToc(); CGS_Progress.push_back(P_cgs);

    for (int cgs_iter = 0; cgs_iter < VTK_Contents.totalBoxElements; cgs_iter++) {
        //Iterations of the Conjugate Gradient Solver here
        vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_rk, d_rk, d_r_norm_old, d_SolverConfig); //r_k dot r_k
        MatrixMul_Omnidirectional_GPU << <numBlocks3D, threadsPerBlock3D, 0, stream[1] >> > (d_temp, d_pk, d_RHS, d_SolverConfig); //temp=A*p_k
        hipDeviceSynchronize();
        vectorDot_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_pk, d_temp, d_temp_scal, d_SolverConfig); //temp_scal = p_k dot temp
        hipDeviceSynchronize();
        divide << <1, 1, 0, stream[0] >> > (d_r_norm_old, d_temp_scal, d_alpha);//alpha = (rk dot rk) / (pk dot A*pk)
        hipDeviceSynchronize(); 

        //Implicit residual update
        scalarVectorMult_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_alpha, d_temp, d_temp, d_SolverConfig); //temp=alphak*temp
        subtractVectors_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[2] >> > (d_rk, d_temp, d_rkp1, d_SolverConfig); //r_k+1=rk-temp (i.e. rk-A*temp)

        hipDeviceSynchronize();
        scalarVectorMult_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_alpha, d_pk, d_temp, d_SolverConfig); //temp = alphak*pk
        hipDeviceSynchronize();
        addVectors_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_PressureField, d_temp, d_PressureField, d_SolverConfig); //xk+1=xk+alphak*pk
        hipDeviceSynchronize();
        //printVector1_GPU << <1, 1 >> > (d_PressureField); hipDeviceSynchronize();
        //printVector2_GPU << <1, 1 >> > (d_PressureField);

        //Explicit residual update
            //MatrixMul_Omnidirectional_GPU << <numBlocks3D, threadsPerBlock3D >> > (d_temp, d_PressureField, d_RHS, d_SolverConfig); //temp=A*x_k+1
            //hipDeviceSynchronize();
            //subtractVectors_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_RHS, d_temp, d_rkp1, d_SolverConfig); //r_k+1=b-A*xk+1
            //hipDeviceSynchronize();

        hipMemcpyAsync(d_rk, d_rkp1, boxArraySize, hipMemcpyDeviceToDevice, stream[0]); //rk=rk+1
        vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_rkp1, d_rkp1, d_r_norm, d_SolverConfig); //r_k+1 dot r_k+1
        hipDeviceSynchronize();

        varfloat r_norm; hipMemcpy(&r_norm, d_r_norm, sizeof(varfloat), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        r_norm = sqrt(r_norm);

        //Handles output to the user
        if (cgs_iter % 10 == 0) {
            if (SolverConfig.Verbose){ mexPrintf("CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e; \n", cgs_iter, r_norm / r_norm_init, r_norm);fflush(stdout);}
        }        
        if (!SolverConfig.Verbose){
            //Even if not verbose, still prints every 10 seconds
            if (ClockToc() > (IdleClock + 10.0)) {
                mexPrintf("CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e; \n", cgs_iter, r_norm / r_norm_init, r_norm);
                fflush(stdout);
                IdleClock += 10.0;
            }
        }

        //Stores iteration info on memory
        P_cgs.Iteration = cgs_iter+1; P_cgs.Residual = r_norm / r_norm_init; P_cgs.TimeSeconds = ClockToc(); CGS_Progress.push_back(P_cgs);

        if ((r_norm / r_norm_init > SolverConfig.solverToleranceRel) && (r_norm > SolverConfig.solverToleranceAbs)) {
            //Only continues if not yet within tolerance
            divide << <1, 1 >> > (d_r_norm, d_r_norm_old, d_beta);//beta = (rk+1 dot rk+1) / (rk dot rk)
            hipDeviceSynchronize(); 
            scalarVectorMult_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_beta, d_pk, d_temp, d_SolverConfig); //temp=beta*pk
            hipDeviceSynchronize();
            addVectors_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_temp, d_rkp1, d_pk, d_SolverConfig); //pk+1=rk+1 + beta*pk 
            hipDeviceSynchronize();
        }
        else {
            if (SolverConfig.Verbose){ mexPrintf("CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e [Converged]\n", cgs_iter, r_norm / r_norm_init, r_norm);} 

            if (isnan(r_norm)) {
                mexPrintf("======== Result was NAN! ========\n");
                mexPrintf("Make sure your coordinate system is correct. This code expects a coordinate in the 'ND grid' format, i.e., dimension order is (x, y) or (x, y, z). DO NOT USE the 'MESHGRID' format to build the arrays in Matlab!\n");
            }
            break;
        }

        if ((r_norm / r_norm_init) > 1e3) {
            //CG is diverging, returns nan
            GPU_FillNan << <numBlocks3D, threadsPerBlock3D, 0, stream[0] >> > (d_PressureField, d_SolverConfig);
            hipDeviceSynchronize();
            mexPrintf("CG Diverged! Exiting.\n", cgs_iter, r_norm / r_norm_init, r_norm);
            break;
        }        
    }

    //Extracts 3D array from GPU Memory
    hipMemcpy(PressureField, d_PressureField, boxArraySize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //Frees memory 
    if (SolverConfig.Verbose){ mexPrintf("==========================================================\n");}
    hipFree(d_PressureField); hipFree(d_RHS); hipFree(d_rk); hipFree(d_rkp1); hipFree(d_pk); hipFree(d_temp);
    hipFree(d_SourceX); hipFree(d_SourceY); hipFree(d_SourceZ);
    hipFree(d_SolverConfig);
    hipFree(d_beta); hipFree(d_alpha); hipFree(d_r_norm); hipFree(d_r_norm_old); hipFree(d_temp_scal);

    //Destroy parallel streams
    for (int i = 0; i < nStreams; ++i) {
        hipStreamDestroy(stream[i]);
    }
}

#pragma endregion

//========CPU OpenMP Functions======
#pragma region
template <typename varfloat>
 void scalarVectorMult_CPU(varfloat* scalar, varfloat* a, varfloat* out, SolverParameters<varfloat>* SP) {
    #pragma omp parallel for
    for (long long i = 0; i < SP->totalBoxElements; i++) {
        out[i] = a[i]* *scalar;
    }
}

template <typename varfloat>
void vectorDot_CPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    //Performs dot product    
    varfloat sum = 0.0;

    #pragma omp parallel for reduction(+:sum)
    for (long long i = 0; i < SP->totalBoxElements; i++) {
        if ((a[i] == a[i]) && (b[i] == b[i])) {
            sum += a[i] * b[i];
            //printf("i=%lld; a=%f, b=%f, tmp=%f, \n", i, a[i], b[i],tmp);
        }
    }
    *out = sum;
}

template <typename varfloat>
void addVectors_CPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
#pragma omp parallel for
    for (long long i = 0; i < SP->totalBoxElements; i++) {
        out[i] = a[i] + b[i];
    }
}

template <typename varfloat>
void subtractVectors_CPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    #pragma omp parallel for
    for (long long i = 0; i < SP->totalBoxElements; i++) {
        out[i] = a[i] - b[i];
    }
}

template <typename varfloat>
void MatrixMul_Omnidirectional_CPU(varfloat* Result, varfloat* PressureField, varfloat* RHS, SolverParameters<varfloat>* SP) {
    //This is the bit of code that performs the matrix multiplication Result=A*x (where A is the weight matrix and x is the PressureField)
    //The RHS of the equation is also provided so we can find the points where we have NAN's    

    if (SP->BoxGridPoints.z == 1) {
        //2D Case
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;

        //dx and dy for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;
        long long zz = 0;

        #pragma omp parallel for
        for (long long xx = 0; xx < SP->BoxGridPoints.x; xx++) {
            for (long long yy = 0; yy < SP->BoxGridPoints.y; yy++) {
                long long idxCenter = xx + GridX * (yy + GridY * zz);

                if ((RHS[idxCenter] != RHS[idxCenter])) {
                    //The RHS here is a nan, so simply makes the result at this point a nan as well
                    Result[idxCenter] = NAN;
                }
                else {
                    long long idx_xp = idxCenter + 1;
                    long long idx_xm = idxCenter - 1;
                    long long idx_yp = idxCenter + GridX;
                    long long idx_ym = idxCenter - GridX;

                    varfloat bxp = ((xx + 1) >= GridX) || (RHS[idx_xp] != RHS[idx_xp]); // isnans exposed as inequalities to reduce the number of registers required (from 112 to 56) [i.e. isnan(X) is the same as X!=X]
                    varfloat byp = ((yy + 1) >= GridY) || (RHS[idx_yp] != RHS[idx_yp]);
                    varfloat bxm = ((xx - 1) < 0) || (RHS[idx_xm] != RHS[idx_xm]);
                    varfloat bym = ((yy - 1) < 0) || (RHS[idx_ym] != RHS[idx_ym]);

                    varfloat rhs_cx = GridDY / (2.0 * (GridDX + GridDY));
                    varfloat rhs_cy = GridDX / (2.0 * (GridDX + GridDY));

                    //Adds the pressure values to right-hand side for this cell 
                    varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym); //Weight for the center coefficient
                    varfloat w_in_1 = 1.0 - w_in;

                    varfloat R = PressureField[idxCenter];
                    R -= bxp ? 0.0 : rhs_cx * PressureField[idx_xp] / w_in_1; //done this way to prevent access outside allocated memory 
                    R -= bxm ? 0.0 : rhs_cx * PressureField[idx_xm] / w_in_1;
                    R -= byp ? 0.0 : rhs_cy * PressureField[idx_yp] / w_in_1;
                    R -= bym ? 0.0 : rhs_cy * PressureField[idx_ym] / w_in_1;
                    Result[idxCenter] = R;
                }
            }
        }
    }
    else {
        //3D Case
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long GridZ = (long long)SP->BoxGridPoints.z;

        //dx and dy and dz for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;
        varfloat GridDZ = SP->GridDelta.z;

        #pragma omp parallel for
        for (long long xx = 0; xx < SP->BoxGridPoints.x; xx++) {
            for (long long yy = 0; yy < SP->BoxGridPoints.y; yy++) {
                for (long long zz = 0; zz < SP->BoxGridPoints.z; zz++) {
                    long long idxCenter = xx + GridX * (yy + GridY * zz);

                    if (RHS[idxCenter] != RHS[idxCenter]) {
                        //The RHS here is a nan, so simply makes the result at this point a nan as well
                        Result[idxCenter] = NAN;
                    }
                    else {
                        long long idx_xp = idxCenter + 1;
                        long long idx_xm = idxCenter - 1;
                        long long idx_yp = idxCenter + GridX;
                        long long idx_ym = idxCenter - GridX;
                        long long idx_zp = idxCenter + GridX * GridY;
                        long long idx_zm = idxCenter - GridX * GridY;

                        varfloat bxp = ((xx + 1) >= GridX) || (RHS[idx_xp] != RHS[idx_xp]); // isnans exposed as inequalities to reduce the number of registers required (from 112 to 80) [i.e. isnan(X) is the same as X!=X]
                        varfloat byp = ((yy + 1) >= GridY) || (RHS[idx_yp] != RHS[idx_yp]);
                        varfloat bzp = ((zz + 1) >= GridZ) || (RHS[idx_zp] != RHS[idx_zp]);
                        varfloat bxm = ((xx - 1) < 0) || (RHS[idx_xm] != RHS[idx_xm]);
                        varfloat bym = ((yy - 1) < 0) || (RHS[idx_ym] != RHS[idx_ym]);
                        varfloat bzm = ((zz - 1) < 0) || (RHS[idx_zm] != RHS[idx_zm]);

                        //Computes the weights for the [n] coefficients
                        varfloat rhs_den = 2.0 * (GridDX * GridDY + GridDX * GridDZ + GridDY * GridDZ);
                        varfloat rhs_cx = (GridDY * GridDZ) / rhs_den;
                        varfloat rhs_cy = (GridDX * GridDZ) / rhs_den;
                        varfloat rhs_cz = (GridDX * GridDY) / rhs_den;

                        //Adds the pressure values to right-hand side for this cell 
                        varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym) + rhs_cz * (bzp + bzm); //Weight for the center coefficient
                        varfloat w_in_1 = 1.0 - w_in;

                        varfloat R = PressureField[idxCenter];
                        R -= bxp ? 0 : rhs_cx * PressureField[idx_xp] / w_in_1; //done this way to prevent access outside allocated memory 
                        R -= bxm ? 0 : rhs_cx * PressureField[idx_xm] / w_in_1;
                        R -= byp ? 0 : rhs_cy * PressureField[idx_yp] / w_in_1;
                        R -= bym ? 0 : rhs_cy * PressureField[idx_ym] / w_in_1;
                        R -= bzp ? 0 : rhs_cz * PressureField[idx_zp] / w_in_1;
                        R -= bzm ? 0 : rhs_cz * PressureField[idx_zm] / w_in_1;
                        Result[idxCenter] = R;
                    }
                }
            }
        }

    }

}

template <typename varfloat>
void UpdateRHS_Vector_CPU(varfloat* PressureField, varfloat* RHS, varfloat* SourceX, varfloat* SourceY, varfloat* SourceZ, SolverParameters<varfloat>* SP) {
    //Computes the right-hand side vector based on the values of the pressures for all cells, considering boundaries, etc.
    
    if (SP->BoxGridPoints.z == 1) {
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;

        //dx and dy for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;

        #pragma omp parallel for
        for (long long xx = 0; xx < SP->BoxGridPoints.x; xx++) {
            for (long long yy = 0; yy < SP->BoxGridPoints.y; yy++) {
                long long zz = 0;
                long long idxCenter = xx + GridX * (yy + GridY * zz);

                if (SourceX[idxCenter] != SourceX[idxCenter]) {
                    //The source value here is a nan, so simply makes the RHS at this point a nan as well
                    RHS[idxCenter] = NAN;
                }
                else {
                        long long idx_xp = idxCenter + 1;
                        long long idx_yp = idxCenter + GridX;
                        long long idx_xm = idxCenter - 1;
                        long long idx_ym = idxCenter - GridX;

                        //Computes the boolean values for each index
                        varfloat bxp = ((xx + 1) >= GridX) || (SourceX[idx_xp] != SourceX[idx_xp]);
                        varfloat byp = ((yy + 1) >= GridY) || (SourceX[idx_yp] != SourceX[idx_yp]);
                        varfloat bxm = ((xx - 1) < 0) || (SourceX[idx_xm] != SourceX[idx_xm]);
                        varfloat bym = ((yy - 1) < 0) || (SourceX[idx_ym] != SourceX[idx_ym]);

                        //Computes the weights for the [n] coefficients
                        //varfloat wxmax = gc->wxx + 2 * gc->wxy; varfloat wymax = gc->wyy + 2 * gc->wxy; //Weights for out-of-bounds conditions
                        varfloat rhs_cx = GridDY / (2.0 * (GridDX + GridDY));
                        varfloat rhs_cy = GridDX / (2.0 * (GridDX + GridDY));

                        varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym); //Weight for the center coefficient
                        varfloat w_in_1 = 1.0 - w_in;

                        //Adds the pressure values to right-hand side for this cell
                        varfloat R = 0.0;
                        R += bxp ? 0.0 : (-rhs_cx * (SourceX[idx_xp] + SourceX[idxCenter]) * (GridDX / 2.0)) / w_in_1;
                        R += bxm ? 0.0 : (rhs_cx * (SourceX[idx_xm] + SourceX[idxCenter]) * (GridDX / 2.0)) / w_in_1;
                        R += byp ? 0.0 : (-rhs_cy * (SourceY[idx_yp] + SourceY[idxCenter]) * (GridDY / 2.0)) / w_in_1;
                        R += bym ? 0.0 : (rhs_cy * (SourceY[idx_ym] + SourceY[idxCenter]) * (GridDY / 2.0)) / w_in_1;
                        RHS[idxCenter] = R;
                    }
            }
        }
    }
    else {
        //3D case
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long GridZ = (long long)SP->BoxGridPoints.z;

        //dx and dy and dz for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;
        varfloat GridDZ = SP->GridDelta.z;

        #pragma omp parallel for
        for (long long xx = 0; xx < SP->BoxGridPoints.x; xx++) {
            for (long long yy = 0; yy < SP->BoxGridPoints.y; yy++) {
                for (long long zz = 0; zz < SP->BoxGridPoints.z; zz++) {
                    long long idxCenter = xx + GridX * (yy + GridY * zz);

                    if (SourceX[idxCenter] != SourceX[idxCenter]) {
                        //The source value here is a nan, so simply makes the RHS at this point a nan as well
                        RHS[idxCenter] = NAN;
                    }
                    else {
                        long long idx_xp = idxCenter + 1;
                        long long idx_xm = idxCenter - 1;
                        long long idx_yp = idxCenter + GridX;
                        long long idx_ym = idxCenter - GridX;
                        long long idx_zp = idxCenter + GridX * GridY;
                        long long idx_zm = idxCenter - GridX * GridY;

                        //Computes the boolean values for each index
                        varfloat bxp = ((xx + 1) >= GridX) || (SourceX[idx_xp] != SourceX[idx_xp]);
                        varfloat bxm = ((xx - 1) < 0) || (SourceX[idx_xm] != SourceX[idx_xm]);
                        varfloat byp = ((yy + 1) >= GridY) || (SourceX[idx_yp] != SourceX[idx_yp]);
                        varfloat bym = ((yy - 1) < 0) || (SourceX[idx_ym] != SourceX[idx_ym]);
                        varfloat bzp = ((zz + 1) >= GridZ) || (SourceX[idx_zp] != SourceX[idx_zp]);
                        varfloat bzm = ((zz - 1) < 0) || (SourceX[idx_zm] != SourceX[idx_zm]);

                        //Computes the weights for the [n] coefficients
                        varfloat rhs_den = 2.0 * (GridDX * GridDY + GridDX * GridDZ + GridDY * GridDZ);
                        varfloat rhs_cx = (GridDY * GridDZ) / rhs_den;
                        varfloat rhs_cy = (GridDX * GridDZ) / rhs_den;
                        varfloat rhs_cz = (GridDX * GridDY) / rhs_den;
            
                        varfloat w_in = rhs_cx * (bxp + bxm) + rhs_cy * (byp + bym) + rhs_cz * (bzp + bzm); //Weight for the center coefficient
                        varfloat w_in_1 = 1.0 - w_in;

                        //Adds the pressure values to right-hand side for this cell   
                        varfloat R = 0.0;
                        R += bxp ? 0.0 : (-rhs_cx * (SourceX[idx_xp] + SourceX[idxCenter]) * GridDX / 2) / w_in_1;
                        R += bxm ? 0.0 : (rhs_cx * (SourceX[idx_xm] + SourceX[idxCenter]) * GridDX / 2) / w_in_1;
                        R += byp ? 0.0 : (-rhs_cy * (SourceY[idx_yp] + SourceY[idxCenter]) * GridDY / 2) / w_in_1;
                        R += bym ? 0.0 : (rhs_cy * (SourceY[idx_ym] + SourceY[idxCenter]) * GridDY / 2) / w_in_1;
                        R += bzp ? 0.0 : (-rhs_cz * (SourceZ[idx_zp] + SourceZ[idxCenter]) * GridDZ / 2) / w_in_1;
                        R += bzm ? 0.0 : (rhs_cz * (SourceZ[idx_zm] + SourceZ[idxCenter]) * GridDZ / 2) / w_in_1;
                        RHS[idxCenter] = R;
                    }
                }
            }
        }
    }
}

template <typename varfloat>
void ConjugateGradientSolver_CPU(varfloat* PressureField, varfloat* RHS, SolverParameters<varfloat> SolverConfig, BoxContents<varfloat> VTK_Contents) {
    // CPU Solver version of the conjugate gradient

    //Allocate memory
    long long boxArraySize = sizeof(varfloat) * VTK_Contents.totalBoxElements;
    varfloat* rk; varfloat* rkp1; varfloat* pk; varfloat* temp;
    rk = (varfloat*)malloc(boxArraySize); rkp1 = (varfloat*)malloc(boxArraySize);
    pk = (varfloat*)malloc(boxArraySize); temp = (varfloat*)malloc(boxArraySize);

    varfloat* beta; varfloat* alpha; varfloat* r_norm; varfloat* r_norm_old; varfloat* temp_scal;
    beta = (varfloat*)malloc(sizeof(varfloat)); alpha = (varfloat*)malloc(sizeof(varfloat)); r_norm = (varfloat*)malloc(sizeof(varfloat));
    r_norm_old = (varfloat*)malloc(sizeof(varfloat)); temp_scal = (varfloat*)malloc(sizeof(varfloat));

    //Start CG solver here [see wikipedia page on Conjugate Gradient to see the steps implemented]
    ClockTic();
    double IdleClock = 0.0; // to keep track of the time so every 10 seconds it still prints something

    UpdateRHS_Vector_CPU(PressureField, RHS, VTK_Contents.SourceFn_Field_X, VTK_Contents.SourceFn_Field_Y, VTK_Contents.SourceFn_Field_Z, &SolverConfig); //b
    MatrixMul_Omnidirectional_CPU(temp, PressureField, RHS, &SolverConfig); //temp=A*x_0
    subtractVectors_CPU(RHS, temp, rk, &SolverConfig); //r_0=b-A*x_0
    memcpy(pk, rk, boxArraySize);
    vectorDot_CPU(rk, rk, r_norm_old, &SolverConfig); //r_k dot r_k

    varfloat r_norm_init; varfloat r_norm_sqrt;
    Progress P_cgs;
    r_norm_init = sqrt(*r_norm_old);
    if (SolverConfig.Verbose){ mexPrintf("Initial Residual Norm=%f\n", r_norm_init);}
    CGS_Progress.clear();
    P_cgs.Iteration = 0; P_cgs.Residual = 1.0f; P_cgs.TimeSeconds = ClockToc(); CGS_Progress.push_back(P_cgs);

    for (int cgs_iter = 0; cgs_iter < VTK_Contents.totalBoxElements; cgs_iter++) {
        //Iterations of the Conjugate Gradient Solver here
        vectorDot_CPU(rk, rk, r_norm_old, &SolverConfig); //r_k dot r_k
        MatrixMul_Omnidirectional_CPU(temp, pk, RHS, &SolverConfig); //temp=A*p_k
        vectorDot_CPU(pk, temp, temp_scal, &SolverConfig); //temp_scal = p_k dot temp
        *alpha = *r_norm_old / *temp_scal;//alpha = (rk dot rk) / (pk dot A*pk)

        //Implicit residual update
        scalarVectorMult_CPU (alpha, temp, temp, &SolverConfig); //temp=alphak*temp
        subtractVectors_CPU (rk, temp, rkp1, &SolverConfig); //r_k+1=rk-temp (i.e. rk-A*temp)

        scalarVectorMult_CPU (alpha, pk, temp, &SolverConfig); //temp = alphak*pk
        addVectors_CPU(PressureField, temp, PressureField, &SolverConfig); //xk+1=xk+alphak*pk

        //Explicit residual update
            //MatrixMul_Omnidirectional_CPU (temp, PressureField, RHS, &SolverConfig); //temp=A*x_k+1
            //subtractVectors_CPU (RHS, temp, rkp1, &SolverConfig); //r_k+1=b-A*xk+1

        memcpy(rk, rkp1, boxArraySize);//rk=rk+1
        vectorDot_CPU (rkp1, rkp1, r_norm, &SolverConfig); //r_k+1 dot r_k+1
        r_norm_sqrt = sqrt(*r_norm);

        if (cgs_iter % 10 == 0) {
            if (SolverConfig.Verbose){ mexPrintf("CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e; \n", cgs_iter, r_norm_sqrt / r_norm_init, r_norm_sqrt);}
        }
        if (!SolverConfig.Verbose){
            //Even if not verbose, still prints every 10 seconds
            if (ClockToc() > (IdleClock + 10.0)) {
                mexPrintf("CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e; \n", cgs_iter, r_norm_sqrt / r_norm_init, r_norm_sqrt);
                IdleClock += 10.0;
            }
        }

        //Stores iteration info on memory
        P_cgs.Iteration = cgs_iter+1; P_cgs.Residual = r_norm_sqrt / r_norm_init; P_cgs.TimeSeconds = ClockToc(); CGS_Progress.push_back(P_cgs);

        if ((r_norm_sqrt / r_norm_init > SolverConfig.solverToleranceRel) && (r_norm_sqrt > SolverConfig.solverToleranceAbs)) {
            //Only continues if not yet within tolerance
            *beta = *r_norm / *r_norm_old;//beta = (rk+1 dot rk+1) / (rk dot rk)
            scalarVectorMult_CPU (beta, pk, temp, &SolverConfig); //temp=beta*pk
            addVectors_CPU(temp, rkp1, pk, &SolverConfig); //pk+1=rk+1 + beta*pk 
        }
        else {
            if (SolverConfig.Verbose){ mexPrintf("CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e [Converged]\n", cgs_iter, r_norm_sqrt / r_norm_init, r_norm_sqrt);}

            if (isnan(r_norm_sqrt)) {
                mexPrintf("======== Result was NAN! ========\n");
                mexPrintf("Make sure your coordinate system is correct. This code expects a coordinate in the 'ND grid' format, i.e., dimension order is (x, y) or (x, y, z). DO NOT USE the 'MESHGRID' format to build the arrays in Matlab!\n");
            }
            return;
        }

        if ((r_norm_sqrt / r_norm_init) > 1e3) {
            //CG is diverging, returns nan
            FillBox(PressureField, ZEROS, SolverConfig);
            mexPrintf("CG Diverged! Exiting.\n");
            break;
        }
    }

    free(rk); free(rkp1);
    free(pk); free(temp);
}

#pragma endregion


//===================Main Matlab function================
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    //Function prototypes:
    //[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz); %Basic form, source term Sx, Sy, Sz are either 2D matrices or 3D matrices. We use **ND grid** format here. Uses default options, and delta=1.
    //[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, delta); %Also provides a grid spacing delta which is the same in all directions.
    //[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, [dx dy dz]); %Provides a grid spacing that is different for x, y, z but is still constant for each direction.
    //[P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, delta, options); %Also provides options as a struct. See below.

    //=====Initializes and checks arguments=====
    //Check for the correct number of input and output arguments
    if (nrhs < 3) {
        mexErrMsgIdAndTxt("OSMODI:mexFunction:invalidNumInputs", "At least 3 input arguments required: [P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, [dx dy dz], options);");
    }
    if (nrhs > 5) {
        mexErrMsgIdAndTxt("OSMODI:mexFunction:invalidNumInputs", "At most 5 input arguments possible: [P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, [dx dy dz], options);");
    }
    if (nlhs != 2) {
        mexErrMsgIdAndTxt("OSMODI:mexFunction:invalidNumOutputs", "Two output arguments required: [P, CGS_Residuals] = OSMODI(Sx, Sy, Sz, [dx dy dz], options);");
    }

    //Ensures all input arguments are single
    if ((!mxIsSingle(prhs[0]) || !mxIsSingle(prhs[1]) || !mxIsSingle(prhs[2])) || ((nrhs > 3) && !mxIsSingle(prhs[3]))) {
        mexErrMsgIdAndTxt("OSMODI:mexFunction:inputNotSingle", "All inputs must be single precision. This code was implemented only in single precision to maximize GPU usage. If the inputs are double, please use: \n [P, CGS_Residuals] = OSMODI(single(Sx), single(Sy), single(Sz), single([dx dy dz]));");
    }

    // Starts filling the arguments
    SolverParameters<float> SolverConfig;
    BoxContents<float> VTK_Contents;
    if (nrhs == 3) {
        SolverConfig.GridDelta = {1.0f, 1.0f, 1.0f};
        VTK_Contents.GridDelta = {1.0f, 1.0f, 1.0f};
    }

    //Ensures Sx, Sy, Sz are of the same size
    int numDimensions1 = mxGetNumberOfDimensions(prhs[0]);
    int numDimensions2 = mxGetNumberOfDimensions(prhs[1]);
    int numDimensions3 = mxGetNumberOfDimensions(prhs[2]);

    if (numDimensions1!=numDimensions2 || numDimensions1!=numDimensions3 || numDimensions2!=numDimensions3) {
        mexErrMsgIdAndTxt("OSMODI:mexFunction:dimensionMismatch", "Sx, Sy and Sz must have the same number of dimensions!");
    }
    if (!(numDimensions1 == 2 || numDimensions1 == 3)) {
        mexErrMsgIdAndTxt("OSMODI:mexFunction:dimensionCount", "Sx, Sy and Sz must be either 2D or 3D matrices!");
    }

    const mwSize *dims1 = mxGetDimensions(prhs[0]);
    const mwSize *dims2 = mxGetDimensions(prhs[1]);
    const mwSize *dims3 = mxGetDimensions(prhs[2]);

    if (numDimensions1==2) {
        if ((dims1[0]!=dims2[0]) || (dims1[0]!=dims3[0]) || (dims2[0]!=dims3[0]) || 
            (dims1[1]!=dims2[1]) || (dims1[1]!=dims3[1]) || (dims2[1]!=dims3[1])) {
            mexErrMsgIdAndTxt("OSMODI:mexFunction:dimensionCountMismatch", "Sx, Sy and Sz must have exactly the same number of dimensions!");
        }
    }
    else if(numDimensions1==3){
        if ((dims1[0]!=dims2[0]) || (dims1[0]!=dims3[0]) || (dims2[0]!=dims3[0]) || 
            (dims1[1]!=dims2[1]) || (dims1[1]!=dims3[1]) || (dims2[1]!=dims3[1]) || 
            (dims1[2]!=dims2[2]) || (dims1[2]!=dims3[2]) || (dims2[2]!=dims3[2])) {
            mexErrMsgIdAndTxt("OSMODI:mexFunction:dimensionCountMismatch", "Sx, Sy and Sz must have exactly the same number of dimensions!");
        }
    }

    //If we're here then Sx, Sy and Sz are 2D or 3D and have the same number of dimensions.
    //Loads the sizes on the configuration variables
    if (numDimensions1==2) {
        SolverConfig.BoxGridPoints = {static_cast<int> (dims1[0]), static_cast<int> (dims1[1]), 1};
        VTK_Contents.BoxGridSize = {static_cast<int> (dims1[0]), static_cast<int> (dims1[1]), 1};
        SolverConfig.totalBoxElements = static_cast<long long> (dims1[0]) * static_cast<long long> (dims1[1]);
        VTK_Contents.totalBoxElements = static_cast<long long> (dims1[0]) * static_cast<long long> (dims1[1]);
    }
    else if(numDimensions1==3){
        SolverConfig.BoxGridPoints = {static_cast<int> (dims1[0]), static_cast<int> (dims1[1]), static_cast<int> (dims1[2])};
        VTK_Contents.BoxGridSize = {static_cast<int> (dims1[0]), static_cast<int> (dims1[1]), static_cast<int> (dims1[2])};
        SolverConfig.totalBoxElements = static_cast<long long> (dims1[0]) * static_cast<long long> (dims1[1]) * static_cast<long long> (dims1[2]);
        VTK_Contents.totalBoxElements = static_cast<long long> (dims1[0]) * static_cast<long long> (dims1[1]) * static_cast<long long> (dims1[2]);
    }
    

    //Loads the actual field pointers to memory
    VTK_Contents.SourceFn_Field_X = (float*) mxGetPr(prhs[0]);
    VTK_Contents.SourceFn_Field_Y = (float*) mxGetPr(prhs[1]);
    VTK_Contents.SourceFn_Field_Z = (float*) mxGetPr(prhs[2]);

    //Fills in the gridDeltas if provided
    if (nrhs > 3) {
        float* gridDeltas = (float*) mxGetPr(prhs[3]);
        int gridDeltasSize = max(static_cast<int>(mxGetM(prhs[3])), static_cast<int>(mxGetN(prhs[3])));

        if (gridDeltasSize == 1){
            SolverConfig.GridDelta = {gridDeltas[0], gridDeltas[0], gridDeltas[0]};
            VTK_Contents.GridDelta = {gridDeltas[0], gridDeltas[0], gridDeltas[0]};        
        } 
        else if (gridDeltasSize >= 3){
            SolverConfig.GridDelta = {gridDeltas[0], gridDeltas[1], gridDeltas[2]};
            VTK_Contents.GridDelta = {gridDeltas[0], gridDeltas[1], gridDeltas[2]};        
        } 
        else if (gridDeltasSize == 2 && numDimensions1 == 2){
            //Unused dimension z
            SolverConfig.GridDelta = {gridDeltas[0], gridDeltas[1], 1.0f};
            VTK_Contents.GridDelta = {gridDeltas[0], gridDeltas[1], 1.0f};        
        } 
        else{
            mexErrMsgIdAndTxt("OSMODI:mexFunction:gridDeltaError", "Wrong size for the grid delta. If the grid is 2D, delta can be any size, but for a 3D grid delta has to be 1-long or 3-long. If delta is longer than 3, only the first 3 elements are considered.");
        }  
    }

    //Fills in the options
    if (nrhs == 5) {
        const mxArray *inputOptions = prhs[4];

        int fieldNo;
        fieldNo = mxGetFieldNumber(inputOptions, "Verbose");
        if (fieldNo != -1){
            const mxArray *mxVebose = mxGetField(inputOptions, 0, "Verbose");
            float verbose = (float) mxGetScalar(mxVebose);
            if (verbose == 1.0f){
                SolverConfig.Verbose = true;
                mexPrintf("SolverConfig.Verbose = TRUE\n");
            }else{
                SolverConfig.Verbose = false;
            }            
        }

        fieldNo = mxGetFieldNumber(inputOptions, "SolverToleranceRel");
        if (fieldNo != -1){
            const mxArray *mxSolverTolRel = mxGetField(inputOptions, 0, "SolverToleranceRel");
            SolverConfig.solverToleranceRel = (float) mxGetScalar(mxSolverTolRel);
            if (SolverConfig.Verbose){
                mexPrintf("SolverConfig.solverToleranceRel = %f\n", SolverConfig.solverToleranceRel);
            }
        }
        
        fieldNo = mxGetFieldNumber(inputOptions, "SolverToleranceAbs");
        if (fieldNo != -1){
            const mxArray *mxSolverTolAbs = mxGetField(inputOptions, 0, "SolverToleranceAbs");
            SolverConfig.solverToleranceAbs = (float) mxGetScalar(mxSolverTolAbs);
            if (SolverConfig.Verbose){
                mexPrintf("SolverConfig.solverToleranceAbs = %f\n", SolverConfig.solverToleranceAbs);
            }
        }
        
        fieldNo = mxGetFieldNumber(inputOptions, "SolverDevice");
        if (fieldNo != -1){
            const mxArray *mxSolverDev = mxGetField(inputOptions, 0, "SolverDevice");
            if (!mxIsChar(mxSolverDev)) {
                mexErrMsgIdAndTxt("OSMODI:mexFunction:deviceMustBeString", "SolverDevice must be a string, either 'GPU' or 'CPU'.");
            }
            else{
                string buffer = mxArrayToString(mxSolverDev);
                if (iequals(buffer, "CPU")){
                    SolverConfig.SolverDevice = CPU;
                    if (SolverConfig.Verbose){
                        mexPrintf("SolverConfig.SolverDevice = CPU\n");
                    }
                }
                else if(iequals(buffer, "GPU")){
                    SolverConfig.SolverDevice = GPU;
                    if (SolverConfig.Verbose){
                        mexPrintf("SolverConfig.SolverDevice = GPU\n");
                    }
                }
                else{
                    mexErrMsgIdAndTxt("OSMODI:mexFunction:deviceError", "SolverDevice must be either 'GPU' or 'CPU'.");
                }
            }            
        }        
    }

    if (SolverConfig.Verbose){
        mexPrintf("Input Box Size = [%d, %d, %d]\n", SolverConfig.BoxGridPoints.x, SolverConfig.BoxGridPoints.y, SolverConfig.BoxGridPoints.z);
        mexPrintf("Total Box Size = %lld\n", SolverConfig.totalBoxElements);
        mexPrintf("Input Box Deltas = [%f, %f, %f]\n", SolverConfig.GridDelta.x, SolverConfig.GridDelta.y, SolverConfig.GridDelta.z);
    }

    //Prepares the output variable for Pressure in memory
    mxArray *PressureOut;
    if (numDimensions1==2) {
        mwSize arrayDim[2] = {dims1[0], dims1[1]};
        PressureOut = mxCreateNumericArray(numDimensions1,arrayDim, mxSINGLE_CLASS, mxREAL);
    }
    else if(numDimensions1==3){
        mwSize arrayDim[3] = {dims1[0], dims1[1], dims1[2]};
        PressureOut = mxCreateNumericArray(numDimensions1,arrayDim, mxSINGLE_CLASS, mxREAL);
    }
    float *PressureField = static_cast<float*>(mxGetData(PressureOut));

    //Initializes the GPU
    bool GPUSuccess;
    GPUSuccess = InitializeGPU(SolverConfig);
    if (!GPUSuccess) {
        mexPrintf("***Error - GPU Not Initialized. Defaulting to CPU code.***");
    }

    //===Starts the solver====
    if(SolverConfig.Verbose){mexPrintf("========Starting up the solver...========\n");}

    ClockTic();
    // Allocate CPU memory for the result and fills with zeros
    FillBox(PressureField, ZEROS, SolverConfig); // Initializes with zeros

    float* RHS; RHS = (float*)malloc(sizeof(float) * VTK_Contents.totalBoxElements);
    FillBox(RHS, ZEROS, SolverConfig); // Initializes with zeros

    //Solves the equations with the solver
    if ((SolverConfig.SolverDevice == GPU) && GPUSuccess) {
        if(SolverConfig.Verbose){mexPrintf("Starting GPU Solver...\n");}
        ConjugateGradientSolver_GPU(PressureField, RHS, SolverConfig, VTK_Contents);
    }
    else {
        if(SolverConfig.Verbose){mexPrintf("Starting CPU Solver...\n");}
        ConjugateGradientSolver_CPU(PressureField, RHS, SolverConfig, VTK_Contents);
    }

    //Outputs the results
    plhs[0] = PressureOut; //Outputs pressure
        
    int nRows = CGS_Progress.size();//Iterates through the contents of the vector CGS_Progress to output the progress as a matrix
    mxArray *CGS_Progress_mx = mxCreateNumericMatrix(nRows, 3, mxSINGLE_CLASS, mxREAL);
    float *CGS_Progress_f = static_cast<float*>(mxGetData(CGS_Progress_mx));
    for (int i=0; i<nRows; i++){
        CGS_Progress_f[0 * nRows + i] = (float) CGS_Progress[i].Iteration;
        CGS_Progress_f[1 * nRows + i] = CGS_Progress[i].Residual;
        CGS_Progress_f[2 * nRows + i] = (float) CGS_Progress[i].TimeSeconds;
    }    
    plhs[1] = CGS_Progress_mx;

    //Happy message at the end
    if (SolverConfig.Verbose){
        mexPrintf("OSMODI completed successfully!\n");
    }
}
