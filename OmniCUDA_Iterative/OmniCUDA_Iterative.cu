#include "hip/hip_runtime.h"
/*
 * © 2024. Triad National Security, LLC. All rights reserved.
 * This program was produced under U.S. Government contract
 * 89233218CNA000001 for Los Alamos National Laboratory (LANL), which is
 * operated by Triad National Security, LLC for the U.S. Department of
 * Energy/National Nuclear Security Administration. All rights in the 
 * program are reserved by Triad National Security, LLC, and the U.S.
 * Department of Energy/National Nuclear Security Administration. The
 * Government is granted for itself and others acting on its behalf a
 * nonexclusive, paid-up, irrevocable worldwide license in this material
 * to reproduce, prepare. derivative works, distribute copies to the
 * public, perform publicly and display publicly, and to permit
 * others to do so.
 *
 * This program is free software: you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your
 * option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program. If not, see <https://www.gnu.org/licenses/>.
 */

//==============================================================
//==============================================================
//====================OMNIDIRECTIONAL CUDA MATRIX SOLVER========
//==============================================================
//==============================================================
//Developed by Fernando Zigunov and John Charonko (2023) - Extreme Fluids Group - Los Alamos National Laboratory
//V00 - This is the ITERATIVE solver.

#pragma once

#include <iostream>
#include <filesystem>
#include <regex>
#include <fstream> 
#include <string>
#include <vector>
#include <iterator>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <algorithm>
#include <sstream>
#include <iomanip>

using namespace std;
using namespace std::filesystem;

#define PI 3.141592653589793
#define _X 0
#define _Y 1
#define _Z 2

#define ZEROS 0
#define ONES 1

#define CPU 0
#define CPU_MULT 1
#define GPU 2

#define BLOCKDIM_VEC 512

clock_t tic; clock_t toc; double timeTask;
clock_t tic2; clock_t toc2; double timeTask2;

typedef double varfloat; //Change this to "float" for single precision or "double" for double precision. Found that "float" is ~5-7% faster (not really all that much)

// ====================Structure Definitions================ 
#pragma region

template <typename varfloat>
struct varfloat3 {
    varfloat x;
    varfloat y;
    varfloat z;
};

template <typename varfloat>
struct SolverParameters {
    varfloat solverToleranceRel = 1e-4; //Error allowed for the solver
    varfloat solverToleranceAbs = 1e-4; //Error allowed for the solver
    varfloat PressureSolverToleranceRel = 1e-4; //Error allowed for the solver
    int3 BoxGridPoints = { 100, 100, 1 }; //Number of grid points in the box
    string BoxInputFile; //Contains the string with the box data file.
    string BoxOutputFile = "Pressure_<frame>.vtk"; //Contains the string with the box data file.
    varfloat3<varfloat> GridDelta; //Delta value for derivative approximation
    varfloat density = 1; //Density of the fluid [metric, kg/m3]
    long long totalBoxElements; //Tracks the size of the box
    int NumberOfIterations = 1; //Number of iterations for the pressure solver
    int CheckpointIterations = 0; //Defines interval between iterations before a new snapshot is saved
    varfloat OverRelaxationFactor = 1; // Defines the over-relaxation factor to improve convergence
} ; // Structure to define the parameters of the solver; initializes to default values

template <typename varfloat>
struct BoxContents {
    varfloat* xCoords; //Stores the x coordinates
    varfloat* yCoords; //Stores the y coordinates
    varfloat* zCoords; //Stores the z coordinates
    varfloat* SourceFn_Field_X; //Stores the Source Function here
    varfloat* SourceFn_Field_Y; //Stores the Source Function here
    varfloat* SourceFn_Field_Z; //Stores the Source Function here
    int3 BoxGridSize; //Number of grid points in the box
    varfloat3<varfloat> GridDelta; //Delta value for derivative approximation
    long long totalBoxElements; //Tracks the size of the box
} ; // Structure to hold the contents of the 3D box 

template <typename varfloat>
struct BoxWeights {
    varfloat* Wxp; //Stores the Source Function here
    varfloat* Wxm; //Stores the Source Function here
    varfloat* Wyp; //Stores the Source Function here
    varfloat* Wym; //Stores the Source Function here
    varfloat* Wzp; //Stores the Source Function here
    varfloat* Wzm; //Stores the Source Function here
} ; // Structure to hold the precomputed weights

template <typename varfloat>
struct gridCoefficients {
    varfloat cxx;
    varfloat cxy;
    varfloat cxz;
    varfloat cyy;
    varfloat cyz;
    varfloat czz;
    varfloat ctot; //total to use as a denominator (considers all angles)

    varfloat wxx; //w coefficients are normalized by ctot
    varfloat wxy;
    varfloat wxz;
    varfloat wyy;
    varfloat wyz;
    varfloat wzz;
} ; // Structure to hold the grid coefficients for each of the neighboring cells

#pragma endregion

// ====================Helper Functions================ 
#pragma region
template <typename varfloat>
__host__ __device__ inline int isnan2(varfloat x)
{
    //Apparently uses less registers than the original isnan, See:
    //https://stackoverflow.com/questions/33922103/is-isnan2-as-fast-as-testing-equality
    return x != x;
}

void ClockTic() {
    //Starts the clock
    tic = clock();
}
void ClockTic2() {
    //Starts the clock
    tic2 = clock();
}

void ClockToc(string Text) {
    //Stops the clock & Prints result
    toc = clock() - tic;
    timeTask = ((double)toc) / CLOCKS_PER_SEC; // in seconds
    printf("%s %f s\n", Text.c_str(), timeTask);
}

void ClockToc2(string Text) {
    //Stops the clock & Prints result
    toc2 = clock() - tic2;
    timeTask2 = ((double)toc2) / CLOCKS_PER_SEC; // in seconds
    printf("%s %f s\n", Text.c_str(), timeTask2);
}

template <typename varfloat>
__host__ __device__ varfloat Norm(varfloat3<varfloat> V) {
    //Computes the vector norm
    return sqrt(V.x * V.x + V.y * V.y + V.z * V.z);
}

template <typename varfloat>
__host__ __device__ varfloat3<varfloat> SubtractVectors(varfloat3<varfloat> A, varfloat3<varfloat> B) {
    //Computes the vector subtraction A-B    
    return { A.x - B.x, A.y - B.y, A.z - B.z };
}

bool iequals(const string& a, const string& b)
{
    //Adapted from answers in
    //https://stackoverflow.com/questions/11635/case-insensitive-string-comparison-in-c
    return std::equal(a.begin(), a.end(),
        b.begin(), b.end(),
        [](char a, char b) {
            return tolower(a) == tolower(b);
        });
}

vector<string> split(const string &text, char delim) {
    //little helper function to split strings, inspired by
    //https://stackoverflow.com/a/7408245/20827864
     vector<string> wordvec;
     istringstream iss(text+delim);
     string word;
     while (getline(iss, word, delim)) {
         wordvec.push_back(word);
     }
     return wordvec;
 }

bool replace(std::string& str, const std::string& from, const std::string& to) {
    //Common utility function, see similar implementations in
    //https://stackoverflow.com/questions/3418231/replace-part-of-a-string-with-another-string
    //and
    //https://stackoverflow.com/questions/5878775/how-to-find-and-replace-string.
    size_t start_pos = str.find(from);
    if (start_pos == std::string::npos)
        return false;
    str.replace(start_pos, from.length(), to);
    return true;
}

void eraseSubStr(std::string& str, const std::string& toErase)
{
    replace(str,toErase,"");
}

void OutputFileName(string& GenericString, int FrameIndex) {
    //Converts the generic input string (i.e., Image<Camera>) into a numbered string (i.e., Image0)
    replace(GenericString, "<Frame>", to_string(FrameIndex));
    replace(GenericString, "<frame>", to_string(FrameIndex));
    return;
}

template <typename varfloat>
bool ParseInputParameterFile(string fileName, SolverParameters<varfloat>& SP) {
    //This function reads the input parameter file, which initializes the solver parameters as well as the file IO.
    //Input file has the following format:
    //PPPPPP XX
    //where PPPPPP is the parameter name, XX is the parameter value. A line break character separates the parameters. 

    ifstream fileRead(fileName, ifstream::in);
    if (!fileRead) {
        printf("Unable to open input file!");
        return false;
    }
    string currentLine;
    int lineNumber = 0;
    while (getline(fileRead, currentLine))
    {
        //Goes through the file, line by line.
        vector<string> words = split(currentLine, ' '); //Splits string into its "words"

        printf("===line #%d: ", lineNumber + 1); //Prints out line for debugging purposes
        cout << currentLine << "===" << endl; //Prints out line for debugging purposes

        //Parses parameters here.
        if (words[0].compare("SP_CGsolverToleranceRel") == 0) {
            printf("Parameter 'SP_CGsolverToleranceRel' identified. \n");
            if (words.size() >= 2) {
                SP.solverToleranceRel = stod(words[1]);
                printf("SolverParameters.SP_CGsolverToleranceRel = %f.\n", SP.solverToleranceRel);
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else if (words[0].compare("SP_CGsolverToleranceAbs") == 0) {
            printf("Parameter 'SP_CGsolverToleranceAbs' identified. \n");
            if (words.size() >= 2) {
                SP.solverToleranceAbs = stod(words[1]);
                printf("SolverParameters.SP_CGsolverToleranceAbs = %f.\n", SP.solverToleranceAbs);
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else if (words[0].compare("SP_PressureSolverToleranceRel") == 0) {
            printf("Parameter 'SP_PressureSolverToleranceRel' identified. \n");
            if (words.size() >= 2) {
                SP.PressureSolverToleranceRel = stod(words[1]);
                printf("SolverParameters.SP_PressureSolverToleranceRel = %f.\n", SP.PressureSolverToleranceRel);
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else if (words[0].compare("SP_CheckpointIterations") == 0) {
            printf("Parameter 'SP_CheckpointIterations' identified. \n");
            if (words.size() >= 2) {
                SP.CheckpointIterations = stoi(words[1]);
                printf("SolverParameters.SP_CheckpointIterations = %f.\n", SP.CheckpointIterations);
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else if (words[0].compare("SP_BoxOutputFile") == 0) {
            printf("Parameter 'SP_BoxOutputFile' identified. \n");
            if (words.size() >= 2) {
                //Defines the box data file as the entire content of this line to account for potential spaces in the file path
                int pos = 17;
                printf(currentLine.substr(pos, currentLine.length() - pos).c_str()); printf("\n");
                SP.BoxOutputFile = currentLine.substr(pos, currentLine.length() - pos);
                printf("SolverParameters.BoxOutputFile = %s.\n", SP.BoxOutputFile.c_str());
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else if (words[0].compare("SP_BoxInputFile") == 0) {
            printf("Parameter 'SP_BoxInputFile' identified. \n");
            if (words.size() >= 2) {
                //Defines the box data file as the entire content of this line to account for potential spaces in the file path
                int pos = 16;
                printf(currentLine.substr(pos, currentLine.length() - pos).c_str()); printf("\n");
                SP.BoxInputFile = currentLine.substr(pos, currentLine.length() - pos);
                printf("SolverParameters.BoxInputFile = %s.\n", SP.BoxInputFile.c_str());
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else if (words[0].compare("SP_NumberOfIterations") == 0) {
            printf("Parameter 'SP_NumberOfIterations' identified. \n");
            if (words.size() >= 2) {
                SP.NumberOfIterations = stoi(words[1]);
                printf("SolverParameters.SP_NumberOfIterations = %f.\n", SP.NumberOfIterations);
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else if (words[0].compare("SP_OverRelaxation") == 0) {
            printf("Parameter 'SP_OverRelaxation' identified. \n");
            if (words.size() >= 2) {
                SP.OverRelaxationFactor = stod(words[1]);
                printf("SolverParameters.SP_OverRelaxation = %f.\n", SP.OverRelaxationFactor);
            }
            else {
                printf("Error in line #%d: Not enough arguments (1 argument required). \n", lineNumber + 1);
            }
        }
        else {
            //Parameter not programmed
            printf("This parameter was not recognized. Ignoring line #%d. \n", lineNumber + 1);
            //printf("Parameter: %s\n", words[0].c_str());
        }

        lineNumber++;
    }

    return true;
}

float ReverseFloat(const float inFloat) {
    //Reverses byte order
    float retVal;
    char* floatToConvert = (char*)&inFloat;
    char* returnFloat = (char*)&retVal;

    // swap the bytes into a temporary buffer
    returnFloat[0] = floatToConvert[3];
    returnFloat[1] = floatToConvert[2];
    returnFloat[2] = floatToConvert[1];
    returnFloat[3] = floatToConvert[0];

    return retVal;
}

template <typename varfloat>
bool ReadVTK(string fileName, BoxContents<varfloat>& VTK_Contents) {
    //This function will read a VTK file produced by vtkwrite in Matlab. 
    //Limitations: 
    //[1] Uses vtk v2.0 and only supports box-like input data with the same number of grid points in the X, Y and Z directions.  
    //[2] There must be two fields inside the VTK: (1) Emission and (2) IOR. If IOR is not to be simulated, fill IOR with ones.
    //[3] Only supports the BINARY file format
    //[4] Only supports RECTILINEAR_GRID as the dataset format
    //[5] Binary data is big-endian

    //Expected format follows https://kitware.github.io/vtk-examples/site/VTKFileFormats/ (Accessed in Jan 2023)
    //# vtk Datafile Version 2.0
    //BINARY
    //DATASET RECTILINEAR_GRID
    //DIMENSIONS xx yy zz (should be the same, xx=yy=zz)
    //X_COORDINATES xx float
    //#&$^%(@*&%@)(#%*&@#_)(*%&@#_#@*%&@#_%& (i.e., binary data for X_COORDINATES)
    //Y_COORDINATES yy float
    //#&$^%(@*&%@)(#%*&@#_)(*%&@#_#@*%&@#_%& (i.e., binary data for Y_COORDINATES)
    //Z_COORDINATES zz float
    //#&$^%(@*&%@)(#%*&@#_)(*%&@#_#@*%&@#_%& (i.e., binary data for Z_COORDINATES)
    //POINT_DATA xx*yy*zz
    //VECTORS U float
    //LOOKUP_TABLE default
    //#&$^%(@*&%@)(#%*&@#_)(*%&@#_#@*%&@#_%& (i.e., binary data for U (velocity))
    //VECTORS DUDT float
    //LOOKUP_TABLE default
    //#&$^%(@*&%@)(#%*&@#_)(*%&@#_#@*%&@#_%& (i.e., binary data for DUDT (velocity time derivative))

    ifstream fileRead(fileName, std::ios::binary);
    if (!fileRead) {
        printf("Unable to open box data file!\n");
        return false;
    }

    string currentLine;

    //Line 1: File header
    printf("Reading Box File %s: Line [1]... (VTK header)\n", fileName.c_str());
    getline(fileRead, currentLine);
    vector<string> words = split(currentLine, ' '); //Splits string into its "words"
    bool isVTK = false;
    for (int i = 0; i < words.size(); i++) {
        /*if ((words[i].compare("vtk") == 0)){
            isVTK = true;
        }*/
        if (iequals(words[i], "vtk")) {
            isVTK = true;
        }
    }
    if (!isVTK) {
        printf("Incompatible file header! Header must contain the word 'vtk'.");
        return false;
    }
    printf("Compatible header VTK found.\n");

    //Line 2: File description (irrelevant)
    printf("Reading Box File %s: Line [2]... (File Description)\n", fileName.c_str()); getline(fileRead, currentLine);

    //Line 3: Binary/ASCII
    printf("Reading Box File %s: Line [3]... (Binary flag)\n", fileName.c_str());
    getline(fileRead, currentLine);
    if (!iequals(currentLine, "binary")) {
        printf("Binary flag is not set! Line [3] should have the text 'BINARY' in it.");
        return false;
    }
    printf("Compatible BINARY flag found.\n");

    //Line 4: Rectilinear grid flag
    printf("Reading Box File %s: Line [4]... (Rectilinear grid flag)\n", fileName.c_str());
    getline(fileRead, currentLine);
    words = split(currentLine, ' '); //Splits string into its "words"
    if (!(iequals(words[0], "dataset") && iequals(words[1], "rectilinear_grid"))) {
        printf("Rectilinear flag is not set! Line [4] should have the text 'DATASET RECTILINEAR_GRID' in it.");
        return false;
    }
    printf("Compatible RECTILINEAR_GRID flag found.\n");

    //Line 5: Data dimensions
    printf("Reading Box File %s: Line [5]... (Data dimensions)\n", fileName.c_str());
    getline(fileRead, currentLine);
    words = split(currentLine, ' '); //Splits string into its "words"
    int WordSize;
    if (!iequals(words[0], "dimensions")) {
        printf("Line [5] should begin with the text 'DIMENSIONS'.");
        return false;
    }
    int xDim; int yDim; int zDim;
    if (words.size() >= 4) {
        xDim = stoi(words[1]); yDim = stoi(words[2]); zDim = stoi(words[3]);
    }
    else {
        printf("Line [5] needs at least 3 arguments (e.g.:'DIMENSIONS 64 64 64').");
        return false;
    }
    VTK_Contents.BoxGridSize.x = xDim;
    VTK_Contents.BoxGridSize.y = yDim;
    VTK_Contents.BoxGridSize.z = zDim;
    printf("Grid dimensions are correctly identified: %d x %d x %d.\n", xDim, yDim, zDim);
    VTK_Contents.totalBoxElements = (long long)xDim * (long long)yDim * (long long)zDim;

    VTK_Contents.SourceFn_Field_X = (varfloat*)malloc(VTK_Contents.totalBoxElements * sizeof(varfloat));
    VTK_Contents.SourceFn_Field_Y = (varfloat*)malloc(VTK_Contents.totalBoxElements * sizeof(varfloat));
    VTK_Contents.SourceFn_Field_Z = (varfloat*)malloc(VTK_Contents.totalBoxElements * sizeof(varfloat));

    //Preallocates memory for 1D arrays for each coordinate
    VTK_Contents.xCoords = (varfloat*)malloc(VTK_Contents.BoxGridSize.x * sizeof(varfloat));
    VTK_Contents.yCoords = (varfloat*)malloc(VTK_Contents.BoxGridSize.y * sizeof(varfloat));
    VTK_Contents.zCoords = (varfloat*)malloc(VTK_Contents.BoxGridSize.z * sizeof(varfloat));

    //Gets the grid spacings from the coordinates
    //---X coordinates
    printf("Reading Box File %s: (X coordinates)\n", fileName.c_str());
    getline(fileRead, currentLine);
    words = split(currentLine, ' '); //Splits string into its "words"
    if (!iequals(words[0], "x_coordinates")) {
        printf("The next line should begin with the text 'X_COORDINATES'.");
        return false;
    }
    float currentVal; char currentCharVal_BigEndian[4]; //float is 4 chars long
    char currentCharVal_LittleEndian[4];
    for (int xx = 0; xx < xDim; xx++) {
        fileRead.read(currentCharVal_BigEndian, 4); //Reads the file. It is generated as a big-endian 4-byte float
        for (int ii = 0; ii < 4; ii++) {
            currentCharVal_LittleEndian[3 - ii] = currentCharVal_BigEndian[ii]; //converts to Little-endian
        }
        memcpy(&currentVal, &currentCharVal_LittleEndian[0], sizeof(currentVal));
        VTK_Contents.xCoords[xx] = (varfloat) currentVal;
        //printf("%f;", VTK_Contents.xCoords[xx]);
    }
    getline(fileRead, currentLine); //skips remaining contents of the line before going to y-coords

    //---Y coordinates
    printf("Reading Box File %s: (Y coordinates)\n", fileName.c_str());
    getline(fileRead, currentLine);
    words = split(currentLine, ' '); //Splits string into its "words"
    if (!iequals(words[0], "y_coordinates")) {
        printf("The next line should begin with the text 'Y_COORDINATES'.");
        return false;
    }
    for (int yy = 0; yy < yDim; yy++) {
        fileRead.read(currentCharVal_BigEndian, 4); //Reads the file. It is generated as a big-endian 4-byte float
        for (int ii = 0; ii < 4; ii++) {
            currentCharVal_LittleEndian[3 - ii] = currentCharVal_BigEndian[ii]; //converts to Little-endian
        }
        memcpy(&currentVal, &currentCharVal_LittleEndian[0], sizeof(currentVal));
        VTK_Contents.yCoords[yy] = (varfloat)currentVal;
        //printf("%f;", VTK_Contents.yCoords[yy]);
    }
    getline(fileRead, currentLine); //skips remaining contents of the line before going to z-coords

    //---Z coordinates
    printf("Reading Box File %s: (Z coordinates)\n", fileName.c_str());
    getline(fileRead, currentLine);
    words = split(currentLine, ' '); //Splits string into its "words"
    if (!iequals(words[0], "z_coordinates")) {
        printf("The next line should begin with the text 'Z_COORDINATES'.");
        return false;
    }
    for (int zz = 0; zz < zDim; zz++) {
        fileRead.read(currentCharVal_BigEndian, 4); //Reads the file. It is generated as a big-endian 4-byte float
        for (int ii = 0; ii < 4; ii++) {
            currentCharVal_LittleEndian[3 - ii] = currentCharVal_BigEndian[ii]; //converts to Little-endian
        }
        memcpy(&currentVal, &currentCharVal_LittleEndian[0], sizeof(currentVal));
        VTK_Contents.zCoords[zz] = (varfloat)currentVal;
        //printf("%f;", VTK_Contents.zCoords[zz]);
    }

    //Computes the grid spacings for each grid direction
    VTK_Contents.GridDelta.x = VTK_Contents.xCoords[1] - VTK_Contents.xCoords[0];
    VTK_Contents.GridDelta.y = VTK_Contents.yCoords[1] - VTK_Contents.yCoords[0];
    if (zDim == 1) {
        VTK_Contents.GridDelta.z = 0;
    }
    else {
        VTK_Contents.GridDelta.z = VTK_Contents.zCoords[1] - VTK_Contents.zCoords[0];
    }
    printf("\nGrid Deltas = [%f;%f;%f]\n\n", VTK_Contents.GridDelta.x, VTK_Contents.GridDelta.y, VTK_Contents.GridDelta.z);

    if (VTK_Contents.GridDelta.x < 0 || VTK_Contents.GridDelta.y < 0 || VTK_Contents.GridDelta.z < 0) {
        //Negative grid spacing means coordinate system is backwards
        printf("\n~~~~~~~~~~~~~Error!! Grid Deltas must all be positive!~~~~~~~~~~~~~~~");
        printf("\nIf you are getting this error, it means the order of the elements in the x, y or z vectors is backwards. Flip the order of the vectors! Also, don't forget to flip the sign of the derivatives in that axis as well, as this code assumes a right-handed coordinate system.");
        printf("\nAborting.");
        abort();
    }

    //Finds the string VECTORS; 
    float Progress;
    for (int vectorPos = 0; vectorPos < 1; vectorPos++) { //Accommodating for multiple vector fields if required
        //Finds the string VECTORS:
        printf("\n Reading Box File %s: Finding string 'VECTORS'...\n", fileName.c_str());

        bool stringFound = false;
        string currentWord = "";
        char currentChar;
        while (!(stringFound || fileRead.eof())) {
            fileRead.get(currentChar);
            //printf("New Character: %c; ", currentChar);

            currentWord.push_back(currentChar);
            if (currentWord.size() > 7) {
                //Only looks at the last 7 characters to try and form the string VECTORS
                currentWord = currentWord.substr(1, currentWord.size() - 1);
            }

            //printf("Current String: %s; \n", currentWord);        
            if (iequals(currentWord, "vectors")) {
                //Found it!
                stringFound = true;
                int A = fileRead.tellg();
                printf("Found string 'VECTORS' at position: %i; ", A);
            }
        }
        if (fileRead.eof() && (vectorPos == 0)) {
            //VECTORS is not in the file!
            printf("Error. The word 'VECTORS' is not present in the file, so the contents of the box can't be read.\n");
            return false;
        }

        string currentVectorField = "";
        bool foundSpace = false; string currentShar;
        fileRead.get(currentChar);
        while (!foundSpace) {
            fileRead.get(currentChar);
            currentShar = currentChar;

            if (iequals(currentShar, " ")) {
                foundSpace = true;
            }
            else {
                currentVectorField.push_back(currentChar);
            }
        }
        printf("Vector field name: '%s'\n", currentWord.c_str());

        if (iequals(currentVectorField, "source")) {
            printf("Field correctly identified as a valid field.\n");
            printf("Populating %s field now...\n", currentVectorField.c_str());
            getline(fileRead, currentLine); //finishes reading the current line (data type is always float)

            //Now we will read actual data. Includes a for loop for this one.        
            long long idx = 0;
            //float currentVal; char currentCharVal_BigEndian[4]; //float is 4 chars long
            //char currentCharVal_LittleEndian[4];
            if (iequals(currentVectorField, "source")) {
                for (int zz = 0; zz < zDim; zz++) {
                    for (int yy = 0; yy < yDim; yy++) {
                        for (int xx = 0; xx < xDim; xx++) {
                            //idx = zz*(xDim*yDim) + yy*(xDim) + xx;
                            idx = xx + xDim * (yy + yDim * (zz));

                            fileRead.read(currentCharVal_BigEndian, 4); //Reads the file. It is generated as a big-endian 4-byte float
                            for (int ii = 0; ii < 4; ii++) {
                                currentCharVal_LittleEndian[3 - ii] = currentCharVal_BigEndian[ii]; //converts to Little-endian
                            }
                            memcpy(&currentVal, &currentCharVal_LittleEndian[0], sizeof(currentVal));
                            VTK_Contents.SourceFn_Field_X[idx] = (varfloat)currentVal;

                            fileRead.read(currentCharVal_BigEndian, 4); //Reads the file. It is generated as a big-endian 4-byte float
                            for (int ii = 0; ii < 4; ii++) {
                                currentCharVal_LittleEndian[3 - ii] = currentCharVal_BigEndian[ii]; //converts to Little-endian
                            }
                            memcpy(&currentVal, &currentCharVal_LittleEndian[0], sizeof(currentVal));
                            VTK_Contents.SourceFn_Field_Y[idx] = (varfloat)currentVal;

                            fileRead.read(currentCharVal_BigEndian, 4); //Reads the file. It is generated as a big-endian 4-byte float
                            for (int ii = 0; ii < 4; ii++) {
                                currentCharVal_LittleEndian[3 - ii] = currentCharVal_BigEndian[ii]; //converts to Little-endian
                            }
                            memcpy(&currentVal, &currentCharVal_LittleEndian[0], sizeof(currentVal));
                            VTK_Contents.SourceFn_Field_Z[idx] = (varfloat)currentVal;

                            if (isnan(VTK_Contents.SourceFn_Field_X[idx]) || isnan(VTK_Contents.SourceFn_Field_Y[idx]) || isnan(VTK_Contents.SourceFn_Field_Z[idx])) {
                                VTK_Contents.SourceFn_Field_X[idx] = NAN;
                                VTK_Contents.SourceFn_Field_Y[idx] = NAN;
                                VTK_Contents.SourceFn_Field_Z[idx] = NAN;
                            }

                        }
                    }
                    Progress = (((float)vectorPos * (float)VTK_Contents.totalBoxElements) + (float)idx) / (1 * (float)VTK_Contents.totalBoxElements);
                    printf("\33[2K\r Loading SOURCE Field... Progress %.2f %%", Progress * 100.0);
                    fflush(stdout);
                }
            }
        }
        else {
            printf("This vector field name was not recognized. Only 'SOURCE' is identified as a valid field. SOURCE = -RHO*DU/DT where DU/DT is the material derivative.\n");
            return false;
        }

    }

    fileRead.close();
    return true;
}

template <typename varfloat>
bool SaveVTK(string fileName, varfloat* fieldOut, SolverParameters<varfloat> SP, BoxContents<varfloat> VTK_Contents) {
    //Writes a VTK file containing the output pressure field computed by the algorithm.
    ofstream fileWrite(fileName, std::ios::binary);

    if (!fileWrite) {
        printf("Error opening the VTK file for output!\n");
        return false;
    }
    string stringOut = "# vtk DataFile Version 2.0\n"; //header
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));

    stringOut = "Pressure Field Computed with Omnidirectional Scheme\n"; //file description
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));

    stringOut = "BINARY\n"; //Binary flag
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));

    stringOut = "DATASET RECTILINEAR_GRID\n"; //DATASET flag
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));

    stringOut = "DIMENSIONS "; stringOut.append(to_string(VTK_Contents.BoxGridSize.x)); stringOut.append(" ");  //Dimensions of data
    stringOut.append(to_string(VTK_Contents.BoxGridSize.y)); stringOut.append(" ");
    stringOut.append(to_string(VTK_Contents.BoxGridSize.z)); stringOut.append("\n");
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));

    //----Coordinates---
    float Coord;
    float* CoordListX; CoordListX = new float[VTK_Contents.BoxGridSize.x];
    float* CoordListY; CoordListY = new float[VTK_Contents.BoxGridSize.y];
    float* CoordListZ; CoordListZ = new float[VTK_Contents.BoxGridSize.z];
    for (int i = 0; i < VTK_Contents.BoxGridSize.x; i++) {
        CoordListX[i] = ReverseFloat((float) VTK_Contents.xCoords[i]); //We need to flip the endianness before writing to file
    }
    for (int i = 0; i < VTK_Contents.BoxGridSize.y; i++) {
        CoordListY[i] = ReverseFloat((float)VTK_Contents.yCoords[i]); //We need to flip the endianness before writing to file
    }
    for (int i = 0; i < VTK_Contents.BoxGridSize.z; i++) {
        CoordListZ[i] = ReverseFloat((float)VTK_Contents.zCoords[i]); //We need to flip the endianness before writing to file
    }

    stringOut = "X_COORDINATES "; stringOut.append(to_string(VTK_Contents.BoxGridSize.x)); stringOut.append(" float\n");
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));
    fileWrite.write((char*)CoordListX, sizeof(float) * VTK_Contents.BoxGridSize.x); //write the binary data    
    stringOut = "\n"; fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char)); //new line character

    stringOut = "Y_COORDINATES "; stringOut.append(to_string(VTK_Contents.BoxGridSize.y)); stringOut.append(" float\n");
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));
    fileWrite.write((char*)CoordListY, sizeof(float) * VTK_Contents.BoxGridSize.y); //write the binary data    
    stringOut = "\n"; fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char)); //new line character

    stringOut = "Z_COORDINATES "; stringOut.append(to_string(VTK_Contents.BoxGridSize.z)); stringOut.append(" float\n");
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));
    fileWrite.write((char*)CoordListZ, sizeof(float) * VTK_Contents.BoxGridSize.z); //write the binary data    
    stringOut = "\n"; fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char)); //new line character

    stringOut = "POINT_DATA "; stringOut.append(to_string(VTK_Contents.totalBoxElements)); stringOut.append("\n");
    fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));
    //----Scalar field---
    stringOut = "SCALARS PRESSURE float\n"; fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));
    stringOut = "LOOKUP_TABLE default\n"; fileWrite.write(stringOut.c_str(), stringOut.size() * sizeof(char));

    //We have to flip endianness of the data prior to writing
    float* fieldToSave;
    fieldToSave = (float*)malloc(VTK_Contents.totalBoxElements * sizeof(float));

    for (int i = 0; i < (VTK_Contents.totalBoxElements); i++) {
        fieldToSave[i] = ReverseFloat((float)fieldOut[i]);
    }
    fileWrite.write((char*)fieldToSave, sizeof(float) * VTK_Contents.totalBoxElements); //write the binary data for the 3D field

    fileWrite.close();
    printf("Successfully saved VTK file!\n");
    free(fieldToSave);

    return true;
}

template <typename varfloat>
void gridCoefficients2D(BoxContents<varfloat>& VTK_Contents, gridCoefficients<varfloat>& gridCoeffs) {
    //This function precomputes the coefficients for the grid in 2D (i.e. a trivial case)
    varfloat dx = 1.0;
    varfloat dy = (varfloat) (VTK_Contents.GridDelta.y / VTK_Contents.GridDelta.x); //Preconditioning by normalizing by x

    varfloat dd = sqrt(dx * dx + dy * dy);
    gridCoeffs.cxx = 2 * (dd - dx);
    gridCoeffs.cxy = dx + dy - dd;
    gridCoeffs.cyy = 2 * (dd - dy);
    gridCoeffs.ctot = 2 * (gridCoeffs.cxx + 4 * gridCoeffs.cxy + gridCoeffs.cyy);
    //printf("c=[%f, %f, %f] %f\n", gridCoeffs.cxx, gridCoeffs.cxy, gridCoeffs.cyy, gridCoeffs.ctot);

    gridCoeffs.wxx = gridCoeffs.cxx / gridCoeffs.ctot;
    gridCoeffs.wxy = gridCoeffs.cxy / gridCoeffs.ctot;
    gridCoeffs.wyy = gridCoeffs.cyy / gridCoeffs.ctot;
}

template <typename varfloat>
varfloat computeProjectionConstant(varfloat dx, varfloat dy, varfloat dz, int Npoints, int j, int k) {
    //Here we will compute the projection constant c_jk for a given pair of faces j,k.
    //j and k are values for each of the axes. The constants are defined in the defines, but X=0, Y=1, Z=2
    varfloat dj, dk, dl, dl2;
    varfloat cjk;
    if (j == k) {
        // Far face integral, j = k
        if (j == _X) {
            dj = dx; dl = dy; dl2 = dz;
        }
        else if (j == _Y) {
            dj = dy; dl = dx; dl2 = dz;
        }
        else if (j == _Z) {
            dj = dz; dl = dx; dl2 = dy;
        }

        cjk = 0; //Integrates here
        varfloat prevIntegrand; varfloat nextIntegrand;
        varfloat alpha1 = atan(dj / dl2); varfloat dphi = PI / (2 * (varfloat)Npoints);
        for (varfloat phi = alpha1; phi <= (PI / 2); phi += dphi) {
            varfloat dl1c = dl * cos(phi);
            varfloat dl1s = dl * sin(phi);
            if (phi == alpha1) {
                prevIntegrand = (PI * (dl2 * dl1s - dj * dl1c) + 2 * (dj * dl1c * atan(dj / dl1s) - dl2 * dl1s * atan(dj / dl1s))) * dphi;
            }
            else {
                nextIntegrand = (PI * (dl2 * dl1s - dj * dl1c) + 2 * (dj * dl1c * atan(dj / dl1s) - dl2 * dl1s * atan(dj / dl1s))) * dphi;
                cjk += (prevIntegrand + nextIntegrand) / 2;
                prevIntegrand = nextIntegrand;
            }
        }
    }
    else {
        // Dihedral face integrals
        if (j == _X) {
            dj = dx;
            if (k == _Y) {
                dk = dy; dl = dz;
            }
            else if (k == _Z) {
                dk = dz; dl = dy;
            }
        }
        else if (j == _Y) {
            dj = dy;
            if (k == _X) {
                dk = dx; dl = dz;
            }
            else if (k == _Z) {
                dk = dz; dl = dx;
            }
        }
        else if (j == _Z) {
            dj = dz;
            if (k == _X) {
                dk = dx; dl = dy;
            }
            else if (k == _Y) {
                dk = dy; dl = dx;
            }
        }
        varfloat I_Ac = 0.5 * dl * dl * atanh((dj * dj * dk * dk) / (2 * dl * dl * dl * dl + 2 * dl * dl * dj * dj + 2 * dl * dl * dk * dk + dj * dj * dk * dk)); //Integral A

        varfloat I_Bc = 0;
        varfloat prevIntegrand = 0; varfloat nextIntegrand;
        varfloat alpha1 = atan(dj / dl); varfloat dphi = PI / (2 * (varfloat)Npoints);
        varfloat dl_2 = dl * dl; varfloat dj_2 = dj * dj;
        for (varfloat phi = 0; phi <= (PI / 2); phi += dphi) {
            varfloat dkc = dk * cos(phi); varfloat dkc_2 = dkc * dkc;
            varfloat dks = dk * sin(phi); varfloat dks_2 = dks * dks;

            if (phi <= alpha1) {
                nextIntegrand = (dl * dks * (atan(dl / dkc) - (dl * dkc) / (dl_2 + dkc_2)) + dkc * dks * (log(dkc / sqrt(dl_2 + dkc_2)) + (dl_2) / (2 * (dl_2 + dkc_2)))) * dphi;
            }
            else {
                nextIntegrand = (dl * dks * (atan(dj / dks) - (dj * dks) / (dj_2 + dks_2)) + dkc * dks * (log(dks / sqrt(dj_2 + dks_2)) + (dj_2) / (2 * (dj_2 + dks_2)))) * dphi;
            }
            I_Bc += (prevIntegrand + nextIntegrand) / 2;
            prevIntegrand = nextIntegrand;
        }

        varfloat I_Cc = 0;
        for (varfloat phi = alpha1; phi <= (PI / 2); phi += dphi) {
            if (phi == alpha1) {
                prevIntegrand = 0.5 * (dj * dk * dk * sin(phi) * (2 * dl * sin(phi) - dj * cos(phi)) / (dj * dj + dk * dk * sin(phi) * sin(phi))) * dphi;
            }
            else {
                nextIntegrand = 0.5 * (dj * dk * dk * sin(phi) * (2 * dl * sin(phi) - dj * cos(phi)) / (dj * dj + dk * dk * sin(phi) * sin(phi))) * dphi;
                I_Cc += (prevIntegrand + nextIntegrand) / 2;
                prevIntegrand = nextIntegrand;
            }
        }

        cjk = I_Ac + I_Bc + I_Cc;
    }
    return cjk;
}

template <typename varfloat>
void gridCoefficients3D(BoxContents<varfloat>& VTK_Contents, gridCoefficients<varfloat>& gridCoeffs, int Npoints) {
    //This function precomputes the coefficients for the grid in 3D (a little more involved)
    varfloat dx = 1.0;
    varfloat dy = VTK_Contents.GridDelta.y / VTK_Contents.GridDelta.x; //Preconditioning by normalizing by x
    varfloat dz = VTK_Contents.GridDelta.z / VTK_Contents.GridDelta.x; //Preconditioning by normalizing by x

    if ((dy == 1.0) && (dz == 1.0)) {
        //equal grid spacing, use precomputed constants to save time
        gridCoeffs.cxx = 0.62776840115648713;
        gridCoeffs.cyy = 0.62776840115648713;
        gridCoeffs.czz = 0.62776840115648713;
        gridCoeffs.cxy = 0.62845603826512653;
        gridCoeffs.cxz = 0.62845603826512653;
        gridCoeffs.cyz = 0.62845603826512653;
    }
    else {
        gridCoeffs.cxx = computeProjectionConstant(dx, dy, dz, Npoints, _X, _X);
        gridCoeffs.cxy = computeProjectionConstant(dx, dy, dz, Npoints, _X, _Y);
        gridCoeffs.cxz = computeProjectionConstant(dx, dy, dz, Npoints, _X, _Z);
        gridCoeffs.cyy = computeProjectionConstant(dx, dy, dz, Npoints, _Y, _Y);
        gridCoeffs.cyz = computeProjectionConstant(dx, dy, dz, Npoints, _Y, _Z);
        gridCoeffs.czz = computeProjectionConstant(dx, dy, dz, Npoints, _Z, _Z);
    }

    gridCoeffs.ctot = 2 * (gridCoeffs.cxx + gridCoeffs.cyy + gridCoeffs.czz + 4 * gridCoeffs.cxy + 4 * gridCoeffs.cxz + 4 * gridCoeffs.cyz);
    //printf("c_tot=%f\n", gridCoeffs.ctot);

    varfloat ctot_ideal = 2 * PI * (dx * dy + dx * dz + dy * dz);
    varfloat coeff_rel_error = abs(gridCoeffs.ctot - ctot_ideal) / ctot_ideal;
    //printf("c_err=%e\n", coeff_rel_error);

    gridCoeffs.wxx = gridCoeffs.cxx / gridCoeffs.ctot;
    gridCoeffs.wxy = gridCoeffs.cxy / gridCoeffs.ctot;
    gridCoeffs.wxz = gridCoeffs.cxz / gridCoeffs.ctot;
    gridCoeffs.wyy = gridCoeffs.cyy / gridCoeffs.ctot;
    gridCoeffs.wyz = gridCoeffs.cyz / gridCoeffs.ctot;
    gridCoeffs.wzz = gridCoeffs.czz / gridCoeffs.ctot;
}


template <typename varfloat>
void FillBox(varfloat* boxToFill, int BoxContents, SolverParameters<varfloat> SP) {
    //Fills the 3D box with one of the objects as dictated by BoxContents
    varfloat xLoc, yLoc, zLoc;
    int idxBox;
    //Reference control volumes
    for (int zz = 0; zz < SP.BoxGridPoints.z; zz++) {
        for (int yy = 0; yy < SP.BoxGridPoints.y; yy++) {
            for (int xx = 0; xx < SP.BoxGridPoints.x; xx++) {
                long long idxBox = (long long)xx + (long long)SP.BoxGridPoints.x * ((long long)yy + (long long)SP.BoxGridPoints.y * ((long long)zz));

                if (BoxContents == ZEROS) {
                    boxToFill[idxBox] = 0.0f;
                }
                else if (BoxContents == ONES) {
                    boxToFill[idxBox] = 1.0f;
                }
                else {
                    boxToFill[idxBox] = 0.0f;
                }
            }
        }
    }
}
#pragma endregion

// ====================GPU CUDA Function Kernels================
#pragma region


template <typename varfloat>
__global__ void printScalar_GPU(varfloat* a) {
    //For debugging purposes
    printf("%f\n", *a);
}

template <typename varfloat>
__global__ void printVector1_GPU(varfloat* v) {
    //For debugging purposes
    printf("%e; ", v[0]);
}

template <typename varfloat>
__global__ void printVector2_GPU(varfloat* v) {
    //For debugging purposes
    printf("%e\n", v[49]);
}

template <typename varfloat>
__global__ void addVectors_GPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    long long idxCenter = (long long)blockIdx.x * (long long)blockDim.x + (long long)threadIdx.x;

    if (idxCenter >= (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }
    out[idxCenter] = a[idxCenter] + b[idxCenter];
}

template <typename varfloat>
__global__ void subtractVectors_GPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    long long idxCenter = (long long)blockIdx.x * (long long)blockDim.x + (long long)threadIdx.x;

    if (idxCenter >= (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }
    out[idxCenter] = a[idxCenter] - b[idxCenter];
}

template <typename varfloat>
__global__ void divide(varfloat* num, varfloat* den, varfloat* out) {
    unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (index_x == 0) {
        *out = *num / *den;
    }
}

template <typename varfloat>
__global__ void vectorDot_GPU_Slow(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    //naive slow implementation
    unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    varfloat tmp = 0.0;
    if (index_x == 0) {
        for (long long i = 0; i < (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z); i++) {
            if (!isnan2(a[i]) && !isnan2(b[i])) {
                tmp += a[i] * b[i];
                //printf("i=%lld; a=%f, b=%f, tmp=%f, \n", i, a[i], b[i],tmp);
            }
        }
        *out = tmp;
    }
}

template <typename varfloat>
__global__ void scalarVectorMult_GPU(varfloat* scalar, varfloat* a, varfloat* out, SolverParameters<varfloat>* SP) {
    unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (index_x < (SP->BoxGridPoints.x * SP->BoxGridPoints.y * SP->BoxGridPoints.z)) {
        out[index_x] = a[index_x] * *scalar;
    }
}


template <typename varfloat>
__global__ void vectorDot_GPU(varfloat* a, varfloat* b, varfloat* out, SolverParameters<varfloat>* SP) {
    __shared__ varfloat shared_tmp[BLOCKDIM_VEC];

    long long index = (long long)blockIdx.x * (long long)blockDim.x + (long long)threadIdx.x;
    long long maxIdx = (long long)SP->BoxGridPoints.x * (long long)SP->BoxGridPoints.y * (long long)SP->BoxGridPoints.z;

    if (index == 0) {
        *out = 0.0; //We need to reset the output variable when we start
    }

    if (index < maxIdx) {
        if (!isnan2(a[index]) || !isnan2(b[index])) {
            shared_tmp[threadIdx.x] = a[index] * b[index]; //Dont norm nans
        }
        else {
            shared_tmp[threadIdx.x] = 0.0;//nans become zeros
        }
    }
    else {
        shared_tmp[threadIdx.x] = 0.0;
    }

    __syncthreads();

    // reduction within block
    unsigned int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            shared_tmp[threadIdx.x] += shared_tmp[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    // atomic add the partial reduction in out
    if (threadIdx.x == 0) {
        atomicAdd(out, shared_tmp[0]);
    }
}

template <typename varfloat>
void Precompute_Weights_Iteration_nPlusOne(BoxWeights<varfloat>& Weights, SolverParameters<varfloat> SP, BoxContents<varfloat> VTK_Contents, gridCoefficients<varfloat> gc) {
    //Precomputes the weights to save time in the GPU computation
    long long GridX = SP.BoxGridPoints.x;
    long long GridY = SP.BoxGridPoints.y;
    long long GridZ = SP.BoxGridPoints.z;

    if (GridZ == 1) {
        //2D Case
        long long zz = 0;
        for (long long yy = 0; yy < GridY; yy++) {
            for (long long xx = 0; xx < GridX; xx++) {
                long long idxCenter = xx + GridX * (yy + GridY * zz);

                long long idx_xpxp = idxCenter + 2;
                long long idx_xmxm = idxCenter - 2;
                long long idx_ypyp = idxCenter + GridX * 2;
                long long idx_ymym = idxCenter - GridX * 2;

                long long idx_xpyp = idxCenter + 1 + GridX;
                long long idx_xpym = idxCenter + 1 - GridX;
                long long idx_xmyp = idxCenter - 1 + GridX;
                long long idx_xmym = idxCenter - 1 - GridX;

                //Computes the boolean values for each index
                varfloat bxpxp = ((xx + 2) >= GridX) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpxp]);
                varfloat bypyp = ((yy + 2) >= GridY) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ypyp]);
                varfloat bxmxm = ((xx - 2) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmxm]);
                varfloat bymym = ((yy - 2) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ymym]);

                varfloat bxpyp = ((xx + 1) >= GridX) || ((yy + 1) >= GridY) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpyp]);
                varfloat bxpym = ((xx + 1) >= GridX) || ((yy - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpym]);
                varfloat bxmyp = ((yy + 1) >= GridY) || ((xx - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmyp]);
                varfloat bxmym = ((xx - 1) < 0) || ((yy - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmym]);

                //Computes the weights for the [n] coefficients
                Weights.Wxp[idxCenter] = (1.0 - bxpxp) * gc.wxx + (2.0 - (bxpyp + bxpym)) * gc.wxy;
                Weights.Wxm[idxCenter] = (1.0 - bxmxm) * gc.wxx + (2.0 - (bxmyp + bxmym)) * gc.wxy;
                Weights.Wyp[idxCenter] = (1.0 - bypyp) * gc.wyy + (2.0 - (bxmyp + bxpyp)) * gc.wxy;
                Weights.Wym[idxCenter] = (1.0 - bymym) * gc.wyy + (2.0 - (bxmym + bxpym)) * gc.wxy;
            }
        }

    }
    else {
        //3D case
        for (long long zz = 0; zz < GridZ; zz++) {
            for (long long yy = 0; yy < GridY; yy++) {
                for (long long xx = 0; xx < GridX; xx++) {
                    long long idxCenter = xx + GridX * (yy + GridY * zz);

                    long long idx_xpxp = idxCenter + 2;
                    long long idx_xmxm = idxCenter - 2;
                    long long idx_ypyp = idxCenter + GridX * 2;
                    long long idx_ymym = idxCenter - GridX * 2;
                    long long idx_zpzp = idxCenter + GridX * GridY * 2;
                    long long idx_zmzm = idxCenter - GridX * GridY * 2;

                    long long idx_xpyp = idxCenter + 1 + GridX;
                    long long idx_xpym = idxCenter + 1 - GridX;
                    long long idx_xpzp = idxCenter + 1 + GridX * GridY;
                    long long idx_xpzm = idxCenter + 1 - GridX * GridY;

                    long long idx_xmyp = idxCenter - 1 + GridX;
                    long long idx_xmym = idxCenter - 1 - GridX;
                    long long idx_xmzp = idxCenter - 1 + GridX * GridY;
                    long long idx_xmzm = idxCenter - 1 - GridX * GridY;

                    long long idx_ypzp = idxCenter + GridX + GridX * GridY;
                    long long idx_ypzm = idxCenter + GridX - GridX * GridY;

                    long long idx_ymzp = idxCenter - GridX + GridX * GridY;
                    long long idx_ymzm = idxCenter - GridX - GridX * GridY;

                    varfloat bxpxp = ((xx + 2) >= GridX) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpxp]);
                    varfloat bxmxm = ((xx - 2) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmxm]);
                    varfloat bypyp = ((yy + 2) >= GridY) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ypyp]);
                    varfloat bymym = ((yy - 2) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ymym]);
                    varfloat bzpzp = ((zz + 2) >= GridZ) || isnan2(VTK_Contents.SourceFn_Field_X[idx_zpzp]);
                    varfloat bzmzm = ((zz - 2) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_zmzm]);

                    varfloat bxpyp = ((xx + 1) >= GridX) || ((yy + 1) >= GridY) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpyp]);
                    varfloat bxpym = ((xx + 1) >= GridX) || ((yy - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpym]);
                    varfloat bxpzp = ((xx + 1) >= GridX) || ((zz + 1) >= GridZ) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpzp]);
                    varfloat bxpzm = ((xx + 1) >= GridX) || ((zz - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xpzm]);

                    varfloat bxmyp = ((xx - 1) < 0) || ((yy + 1) >= GridY) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmyp]);
                    varfloat bxmym = ((xx - 1) < 0) || ((yy - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmym]);
                    varfloat bxmzp = ((xx - 1) < 0) || ((zz + 1) >= GridZ) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmzp]);
                    varfloat bxmzm = ((xx - 1) < 0) || ((zz - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_xmzm]);

                    varfloat bypzp = ((yy + 1) >= GridY) || ((zz + 1) >= GridZ) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ypzp]);
                    varfloat bypzm = ((yy + 1) >= GridY) || ((zz - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ypzm]);

                    varfloat bymzp = ((yy - 1) < 0) || ((zz + 1) >= GridZ) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ymzp]);
                    varfloat bymzm = ((yy - 1) < 0) || ((zz - 1) < 0) || isnan2(VTK_Contents.SourceFn_Field_X[idx_ymzm]);

                    //Computes the weights for the [n+1] coefficients                    
                    Weights.Wxp[idxCenter] = (1 - bxpxp) * gc.wxx + (2 - (bxpyp + bxpym)) * gc.wxy + (2 - (bxpzp + bxpzm)) * gc.wxz;
                    Weights.Wxm[idxCenter] = (1 - bxmxm) * gc.wxx + (2 - (bxmyp + bxmym)) * gc.wxy + (2 - (bxmzp + bxmzm)) * gc.wxz;
                    Weights.Wyp[idxCenter] = (1 - bypyp) * gc.wyy + (2 - (bxmyp + bxpyp)) * gc.wxy + (2 - (bypzm + bypzp)) * gc.wyz;
                    Weights.Wym[idxCenter] = (1 - bymym) * gc.wyy + (2 - (bxmym + bxpym)) * gc.wxy + (2 - (bymzm + bymzp)) * gc.wyz;
                    Weights.Wzp[idxCenter] = (1 - bzpzp) * gc.wzz + (2 - (bxmzp + bxpzp)) * gc.wxz + (2 - (bymzp + bypzp)) * gc.wyz;
                    Weights.Wzm[idxCenter] = (1 - bzmzm) * gc.wzz + (2 - (bxmzm + bxpzm)) * gc.wxz + (2 - (bymzm + bypzm)) * gc.wyz;
                }
            }
        }
    }

}

template <typename varfloat>
__global__ void MatrixMul_Omnidirectional_GPU(varfloat* Result, varfloat* PressureField, varfloat* RHS, SolverParameters<varfloat>* SP, gridCoefficients<varfloat>* gc, varfloat* Wxp, varfloat* Wxm, varfloat* Wyp, varfloat* Wym, varfloat* Wzp, varfloat* Wzm) {
    //This is the bit of code that performs the matrix multiplication Result=A*x (where A is the weight matrix and x is the PressureField)
    //The RHS of the equation is also provided so we can find the points where we have NAN's
    long long xx = blockIdx.x * blockDim.x + threadIdx.x;
    long long yy = blockIdx.y * blockDim.y + threadIdx.y;
    long long zz = blockIdx.z * blockDim.z + threadIdx.z;

    if ((xx >= SP->BoxGridPoints.x) || (yy >= SP->BoxGridPoints.y) || (zz >= SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }

    if (SP->BoxGridPoints.z == 1) {
        //2D Case
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;

        long long idxCenter = xx + GridX * (yy + GridY * zz);

        if ((RHS[idxCenter] != RHS[idxCenter])) {
            //The RHS here is a nan, so simply makes the result at this point a nan as well
            Result[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_xm = idxCenter - 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_ym = idxCenter - GridX;

            varfloat bxp = ((xx + 1) >= GridX) || (RHS[idx_xp] != RHS[idx_xp]); // isnans exposed as inequalities to reduce the number of registers required (from 112 to 56) [i.e. isnan(X) is the same as X!=X]
            varfloat byp = ((yy + 1) >= GridY) || (RHS[idx_yp] != RHS[idx_yp]);
            varfloat bxm = ((xx - 1) < 0) || (RHS[idx_xm] != RHS[idx_xm]);
            varfloat bym = ((yy - 1) < 0) || (RHS[idx_ym] != RHS[idx_ym]);

            //Adds the pressure values to right-hand side for this cell 
            varfloat R = PressureField[idxCenter];
            R -= bxp ? 0.0 : Wxp[idxCenter] * PressureField[idx_xp]; //done this way to prevent access outside allocated memory 
            R -= bxm ? 0.0 : Wxm[idxCenter] * PressureField[idx_xm];
            R -= byp ? 0.0 : Wyp[idxCenter] * PressureField[idx_yp];
            R -= bym ? 0.0 : Wym[idxCenter] * PressureField[idx_ym];
            Result[idxCenter] = R;
        }
    }
    else {
        //3D Case
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long GridZ = (long long)SP->BoxGridPoints.z;

        long long idxCenter = xx + GridX * (yy + GridY * zz);

        if (RHS[idxCenter] != RHS[idxCenter]) {
            //The RHS here is a nan, so simply makes the result at this point a nan as well
            Result[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_xm = idxCenter - 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_ym = idxCenter - GridX;
            long long idx_zp = idxCenter + GridX * GridY;
            long long idx_zm = idxCenter - GridX * GridY;

            varfloat bxp = ((xx + 1) >= GridX) || (RHS[idx_xp] != RHS[idx_xp]); // isnans exposed as inequalities to reduce the number of registers required (from 112 to 80) [i.e. isnan(X) is the same as X!=X]
            varfloat byp = ((yy + 1) >= GridY) || (RHS[idx_yp] != RHS[idx_yp]);
            varfloat bzp = ((zz + 1) >= GridZ) || (RHS[idx_zp] != RHS[idx_zp]);
            varfloat bxm = ((xx - 1) < 0) || (RHS[idx_xm] != RHS[idx_xm]);
            varfloat bym = ((yy - 1) < 0) || (RHS[idx_ym] != RHS[idx_ym]);
            varfloat bzm = ((zz - 1) < 0) || (RHS[idx_zm] != RHS[idx_zm]);

            //Adds the pressure values to right-hand side for this cell 
            varfloat R = PressureField[idxCenter];
            R -= bxp ? 0 : Wxp[idxCenter] * PressureField[idx_xp]; //done this way to prevent access outside allocated memory 
            R -= bxm ? 0 : Wxm[idxCenter] * PressureField[idx_xm];
            R -= byp ? 0 : Wyp[idxCenter] * PressureField[idx_yp];
            R -= bym ? 0 : Wym[idxCenter] * PressureField[idx_ym];
            R -= bzp ? 0 : Wzp[idxCenter] * PressureField[idx_zp];
            R -= bzm ? 0 : Wzm[idxCenter] * PressureField[idx_zm];
            Result[idxCenter] = R;
        }

    }

}

template <typename varfloat>
__global__ void UpdateRHS_Vector_GPU(varfloat* PressureField, varfloat* RHS, varfloat* SourceX, varfloat* SourceY, varfloat* SourceZ, SolverParameters<varfloat>* SP, gridCoefficients<varfloat>* gc, varfloat* Wxp, varfloat* Wxm, varfloat* Wyp, varfloat* Wym, varfloat* Wzp, varfloat* Wzm) {
    //Computes the right-hand side vector based on the values of the pressures for all cells, considering boundaries, etc.
    long long xx = blockIdx.x * blockDim.x + threadIdx.x;
    long long yy = blockIdx.y * blockDim.y + threadIdx.y;
    long long zz = blockIdx.z * blockDim.z + threadIdx.z;

    if ((xx >= SP->BoxGridPoints.x) || (yy >= SP->BoxGridPoints.y) || (zz >= SP->BoxGridPoints.z)) {
        //Idles if voxels beyond volume size
        return;
    }

    if (SP->BoxGridPoints.z == 1) {
        //Finds the indices for each of the adjacent cells and their neighbors
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long idxCenter = xx + GridX * (yy + GridY * zz);

        //dx and dy for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;

        if (SourceX[idxCenter] != SourceX[idxCenter]) {
            //The source value here is a nan, so simply makes the RHS at this point a nan as well
            RHS[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_xm = idxCenter - 1;
            long long idx_ym = idxCenter - GridX;

            //Computes the boolean values for each index
            varfloat bxp = ((xx + 1) >= GridX) || (SourceX[idx_xp] != SourceX[idx_xp]);
            varfloat byp = ((yy + 1) >= GridY) || (SourceX[idx_yp] != SourceX[idx_yp]);
            varfloat bxm = ((xx - 1) < 0) || (SourceX[idx_xm] != SourceX[idx_xm]);
            varfloat bym = ((yy - 1) < 0) || (SourceX[idx_ym] != SourceX[idx_ym]);

            //Computes the weights for the [n] coefficients
            varfloat wxmax = gc->wxx + 2 * gc->wxy; varfloat wymax = gc->wyy + 2 * gc->wxy; //Weights for out-of-bounds conditions
            varfloat rhs_cx = GridDY / (2.0 * (GridDX + GridDY));
            varfloat rhs_cy = GridDX / (2.0 * (GridDX + GridDY));

            //Adds the pressure values to right-hand side for this cell
            varfloat R = 0.0;
            R += bxp ? 0.0 : ((wxmax - Wxp[idxCenter]) * PressureField[idx_xp] - rhs_cx * (SourceX[idx_xp] + SourceX[idxCenter]) * (GridDX / 2.0));
            R += bxm ? 0.0 : ((wxmax - Wxm[idxCenter]) * PressureField[idx_xm] + rhs_cx * (SourceX[idx_xm] + SourceX[idxCenter]) * (GridDX / 2.0));
            R += byp ? 0.0 : ((wymax - Wyp[idxCenter]) * PressureField[idx_yp] - rhs_cy * (SourceY[idx_yp] + SourceY[idxCenter]) * (GridDY / 2.0));
            R += bym ? 0.0 : ((wymax - Wym[idxCenter]) * PressureField[idx_ym] + rhs_cy * (SourceY[idx_ym] + SourceY[idxCenter]) * (GridDY / 2.0));

            varfloat w_in = wxmax * (bxp + bxm) + wymax * (byp + bym); //Weight for the center coefficient
            R += w_in * PressureField[idxCenter];
            RHS[idxCenter] = R;

        }
    }
    else {
        //3D case
        long long GridX = (long long)SP->BoxGridPoints.x;
        long long GridY = (long long)SP->BoxGridPoints.y;
        long long GridZ = (long long)SP->BoxGridPoints.z;
        long long idxCenter = xx + GridX * (yy + GridY * zz);

        //dx and dy and dz for the grid
        varfloat GridDX = SP->GridDelta.x;
        varfloat GridDY = SP->GridDelta.y;
        varfloat GridDZ = SP->GridDelta.z;

        if (SourceX[idxCenter] != SourceX[idxCenter]) {
            //The source value here is a nan, so simply makes the RHS at this point a nan as well
            RHS[idxCenter] = NAN;
        }
        else {
            long long idx_xp = idxCenter + 1;
            long long idx_xm = idxCenter - 1;
            long long idx_yp = idxCenter + GridX;
            long long idx_ym = idxCenter - GridX;
            long long idx_zp = idxCenter + GridX * GridY;
            long long idx_zm = idxCenter - GridX * GridY;

            //Computes the boolean values for each index
            varfloat bxp = ((xx + 1) >= GridX) || (SourceX[idx_xp] != SourceX[idx_xp]);
            varfloat bxm = ((xx - 1) < 0) || (SourceX[idx_xm] != SourceX[idx_xm]);
            varfloat byp = ((yy + 1) >= GridY) || (SourceX[idx_yp] != SourceX[idx_yp]);
            varfloat bym = ((yy - 1) < 0) || (SourceX[idx_ym] != SourceX[idx_ym]);
            varfloat bzp = ((zz + 1) >= GridZ) || (SourceX[idx_zp] != SourceX[idx_zp]);
            varfloat bzm = ((zz - 1) < 0) || (SourceX[idx_zm] != SourceX[idx_zm]);

            //Computes the weights for the [n] coefficients
            varfloat wxmax = gc->wxx + 2 * gc->wxy + 2 * gc->wxz; varfloat wymax = gc->wyy + 2 * gc->wxy + 2 * gc->wyz; varfloat wzmax = gc->wzz + 2 * gc->wxz + 2 * gc->wyz;//Weights for out-of-bounds conditions
            varfloat rhs_den = 2.0 * (GridDX * GridDY + GridDX * GridDZ + GridDY * GridDZ);
            varfloat rhs_cx = (GridDY * GridDZ) / rhs_den;
            varfloat rhs_cy = (GridDX * GridDZ) / rhs_den;
            varfloat rhs_cz = (GridDX * GridDY) / rhs_den;

            //Adds the pressure values to right-hand side for this cell   
            varfloat R = 0.0;
            R += bxp ? 0.0 : ((wxmax - Wxp[idxCenter]) * PressureField[idx_xp] - rhs_cx * (SourceX[idx_xp] + SourceX[idxCenter]) * GridDX / 2);
            R += bxm ? 0.0 : ((wxmax - Wxm[idxCenter]) * PressureField[idx_xm] + rhs_cx * (SourceX[idx_xm] + SourceX[idxCenter]) * GridDX / 2);
            R += byp ? 0.0 : ((wymax - Wyp[idxCenter]) * PressureField[idx_yp] - rhs_cy * (SourceY[idx_yp] + SourceY[idxCenter]) * GridDY / 2);
            R += bym ? 0.0 : ((wymax - Wym[idxCenter]) * PressureField[idx_ym] + rhs_cy * (SourceY[idx_ym] + SourceY[idxCenter]) * GridDY / 2);
            R += bzp ? 0.0 : ((wzmax - Wzp[idxCenter]) * PressureField[idx_zp] - rhs_cz * (SourceZ[idx_zp] + SourceZ[idxCenter]) * GridDZ / 2);
            R += bzm ? 0.0 : ((wzmax - Wzm[idxCenter]) * PressureField[idx_zm] + rhs_cz * (SourceZ[idx_zm] + SourceZ[idxCenter]) * GridDZ / 2);

            varfloat w_in = wxmax * (bxp + bxm) + wymax * (byp + bym) + wzmax * (bzp + bzm); //Weight for the center coefficient
            R += w_in * PressureField[idxCenter];
            RHS[idxCenter] = R;
        }
    }
}

template <typename varfloat>
void ConjugateGradientSolver_GPU(varfloat* PressureField, varfloat* RHS, BoxWeights<varfloat> Weights, SolverParameters<varfloat> SolverConfig, BoxContents<varfloat> VTK_Contents, gridCoefficients<varfloat> gridCoeffs, string OutputFileName, ofstream* csvOutput) {
    // Allocate GPU memory for source field and pressure field
    //Creates concurrent streams so processing can occur in parallel
    const int nStreams = 4;
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; ++i) {
        hipStreamCreate(&stream[i]);
    }

    //Memory allocation
    long long boxArraySize = sizeof(varfloat) * VTK_Contents.totalBoxElements;
    varfloat* d_PressureField; varfloat* d_RHS; varfloat* d_PressureFieldOld;
    varfloat* d_rk; varfloat* d_rkp1; varfloat* d_pk; varfloat* d_temp;
    hipMalloc(&d_PressureField, boxArraySize); hipMalloc(&d_RHS, boxArraySize); hipMalloc(&d_PressureFieldOld, boxArraySize);
    hipMalloc(&d_rk, boxArraySize); hipMalloc(&d_rkp1, boxArraySize);
    hipMalloc(&d_pk, boxArraySize); hipMalloc(&d_temp, boxArraySize);

    hipMemcpyAsync(d_PressureField, PressureField, boxArraySize, hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_RHS, RHS, boxArraySize, hipMemcpyHostToDevice, stream[1]);

    //Allocate GPU memory for the weights
    varfloat* d_Wxp; varfloat* d_Wxm; varfloat* d_Wyp; varfloat* d_Wym; varfloat* d_Wzp; varfloat* d_Wzm;
    hipMalloc(&d_Wxp, boxArraySize); hipMalloc(&d_Wxm, boxArraySize); hipMalloc(&d_Wyp, boxArraySize); hipMalloc(&d_Wym, boxArraySize); hipMalloc(&d_Wzp, boxArraySize); hipMalloc(&d_Wzm, boxArraySize);
    hipMemcpyAsync(d_Wxp, Weights.Wxp, boxArraySize, hipMemcpyHostToDevice, stream[2]); hipMemcpyAsync(d_Wxm, Weights.Wxm, boxArraySize, hipMemcpyHostToDevice, stream[3]);
    hipMemcpyAsync(d_Wyp, Weights.Wyp, boxArraySize, hipMemcpyHostToDevice, stream[0]); hipMemcpyAsync(d_Wym, Weights.Wym, boxArraySize, hipMemcpyHostToDevice, stream[1]);
    hipMemcpyAsync(d_Wzp, Weights.Wzp, boxArraySize, hipMemcpyHostToDevice, stream[2]); hipMemcpyAsync(d_Wzm, Weights.Wzm, boxArraySize, hipMemcpyHostToDevice, stream[3]);

    //Allocate GPU memory for the source terms
    varfloat* d_SourceX; varfloat* d_SourceY; varfloat* d_SourceZ;
    hipMalloc(&d_SourceX, boxArraySize); hipMalloc(&d_SourceY, boxArraySize); hipMalloc(&d_SourceZ, boxArraySize);
    hipMemcpyAsync(d_SourceX, VTK_Contents.SourceFn_Field_X, boxArraySize, hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_SourceY, VTK_Contents.SourceFn_Field_Y, boxArraySize, hipMemcpyHostToDevice, stream[1]);
    hipMemcpyAsync(d_SourceZ, VTK_Contents.SourceFn_Field_Z, boxArraySize, hipMemcpyHostToDevice, stream[2]);

    //Allocates auxiliary variables
    gridCoefficients<varfloat>* d_gridCoeffs;
    hipMalloc(&d_gridCoeffs, sizeof(gridCoefficients<varfloat>));
    hipMemcpyAsync(d_gridCoeffs, &gridCoeffs, sizeof(gridCoefficients<varfloat>), hipMemcpyHostToDevice, stream[0]);

    SolverParameters<varfloat>* d_SolverConfig;
    hipMalloc(&d_SolverConfig, sizeof(SolverParameters<varfloat>));
    hipMemcpyAsync(d_SolverConfig, &SolverConfig, sizeof(SolverParameters<varfloat>), hipMemcpyHostToDevice, stream[1]);

    //Allocates scalars
    varfloat* d_beta; varfloat* d_alpha; varfloat* d_r_norm; varfloat* d_r_norm_old; varfloat* d_temp_scal; varfloat* d_over_relax;
    hipMalloc((void**)&d_beta, sizeof(varfloat)); hipMalloc((void**)&d_alpha, sizeof(varfloat)); hipMalloc((void**)&d_r_norm, sizeof(varfloat));
    hipMalloc((void**)&d_r_norm_old, sizeof(varfloat)); hipMalloc((void**)&d_temp_scal, sizeof(varfloat)); hipMalloc((void**)&d_over_relax, sizeof(varfloat));

    hipMemcpy(d_over_relax, &SolverConfig.OverRelaxationFactor, sizeof(varfloat), hipMemcpyHostToDevice);

    dim3 threadsPerBlock3D; dim3 numBlocks3D; //3D for the matrix kernel
    if (VTK_Contents.BoxGridSize.z == 1) {
        threadsPerBlock3D = dim3(16, 16, 1);
        numBlocks3D = dim3(ceil(VTK_Contents.BoxGridSize.x / (varfloat)threadsPerBlock3D.x), ceil(VTK_Contents.BoxGridSize.y / (varfloat)threadsPerBlock3D.y), 1);
    }
    else {
        threadsPerBlock3D = dim3(4, 8, 8);
        numBlocks3D = dim3(ceil(VTK_Contents.BoxGridSize.x / (varfloat)threadsPerBlock3D.x), ceil(VTK_Contents.BoxGridSize.y / (varfloat)threadsPerBlock3D.y), ceil(VTK_Contents.BoxGridSize.z / (varfloat)threadsPerBlock3D.z));
    }
    dim3 threadsPerBlock1D = dim3(BLOCKDIM_VEC, 1, 1); //1D for the vector kernels
    dim3 numBlocks1D = dim3(ceil(VTK_Contents.totalBoxElements / (varfloat)threadsPerBlock1D.x), 1, 1);

    //Outer pressure loop for solver
    varfloat first_r_norm_init = 0.0;
    varfloat p_norm = 1.0; varfloat delta_p_norm = 1.0; //to assess convergence
    for (int pressureIterations = 0; pressureIterations < SolverConfig.NumberOfIterations; pressureIterations++) {
        ClockTic();
        //=====Updates RHS terms=====
        hipDeviceSynchronize();
        UpdateRHS_Vector_GPU << <numBlocks3D, threadsPerBlock3D, 0, stream[0] >> > (d_PressureField, d_RHS, d_SourceX, d_SourceY, d_SourceZ, d_SolverConfig, d_gridCoeffs, d_Wxp, d_Wxm, d_Wyp, d_Wym, d_Wzp, d_Wzm); //temp=A*x_0

        //=====Starts CG solver computations=====
        hipDeviceSynchronize();
        MatrixMul_Omnidirectional_GPU << <numBlocks3D, threadsPerBlock3D, 0, stream[0] >> > (d_temp, d_PressureField, d_RHS, d_SolverConfig, d_gridCoeffs, d_Wxp, d_Wxm, d_Wyp, d_Wym, d_Wzp, d_Wzm); //temp=A*x_0
        hipDeviceSynchronize();
        subtractVectors_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_RHS, d_temp, d_rk, d_SolverConfig); //r_0=b-A*x_0

        hipDeviceSynchronize();
        hipMemcpyAsync(d_pk, d_rk, boxArraySize, hipMemcpyDeviceToDevice, stream[0]); //p_0=r_0
        vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_rk, d_rk, d_r_norm_old, d_SolverConfig); //r_k dot r_kvarfloat r_norm; 
        hipDeviceSynchronize();

        varfloat r_norm_init;
        hipMemcpy(&r_norm_init, d_r_norm_old, sizeof(varfloat), hipMemcpyDeviceToHost); // initial residual norm
        r_norm_init = sqrt(r_norm_init);
        if (pressureIterations == 0) {
            first_r_norm_init = r_norm_init;
        }

        printf("Initial Residual Norm=%f\n", r_norm_init);
        for (int cgs_iter = 0; cgs_iter < VTK_Contents.totalBoxElements; cgs_iter++) {
            //Iterations of the Conjugate Gradient Solver here
            vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_rk, d_rk, d_r_norm_old, d_SolverConfig); //r_k dot r_k
            MatrixMul_Omnidirectional_GPU << <numBlocks3D, threadsPerBlock3D, 0, stream[1] >> > (d_temp, d_pk, d_RHS, d_SolverConfig, d_gridCoeffs, d_Wxp, d_Wxm, d_Wyp, d_Wym, d_Wzp, d_Wzm); //temp=A*p_k
            hipDeviceSynchronize();
            vectorDot_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_pk, d_temp, d_temp_scal, d_SolverConfig); //temp_scal = p_k dot temp
            hipDeviceSynchronize();
            divide << <1, 1, 0, stream[0] >> > (d_r_norm_old, d_temp_scal, d_alpha);//alpha = (rk dot rk) / (pk dot A*pk)

            //Implicit residual update
            scalarVectorMult_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_alpha, d_temp, d_temp, d_SolverConfig); //temp=alphak*temp
            subtractVectors_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[2] >> > (d_rk, d_temp, d_rkp1, d_SolverConfig); //r_k+1=rk-temp (i.e. rk-A*temp)

            hipDeviceSynchronize();
            scalarVectorMult_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_alpha, d_pk, d_temp, d_SolverConfig); //temp = alphak*pk
            hipDeviceSynchronize();
            addVectors_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[0] >> > (d_PressureField, d_temp, d_PressureField, d_SolverConfig); //xk+1=xk+alphak*pk
            hipDeviceSynchronize();
            //printVector1_GPU << <1, 1 >> > (d_PressureField); hipDeviceSynchronize();
            //printVector2_GPU << <1, 1 >> > (d_PressureField);

            //Explicit residual update
                //MatrixMul_Omnidirectional_GPU << <numBlocks3D, threadsPerBlock3D >> > (d_temp, d_PressureField, d_RHS, d_SolverConfig, d_gridCoeffs, d_Wxp, d_Wxm, d_Wyp, d_Wym, d_Wzp, d_Wzm); //temp=A*x_k+1
                //hipDeviceSynchronize();
                //subtractVectors_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_RHS, d_temp, d_rkp1, d_SolverConfig); //r_k+1=b-A*xk+1
                //hipDeviceSynchronize();

            hipMemcpyAsync(d_rk, d_rkp1, boxArraySize, hipMemcpyDeviceToDevice, stream[0]); //rk=rk+1
            vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_rkp1, d_rkp1, d_r_norm, d_SolverConfig); //r_k+1 dot r_k+1
            hipDeviceSynchronize();

            varfloat r_norm; hipMemcpy(&r_norm, d_r_norm, sizeof(varfloat), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            r_norm = sqrt(r_norm);

            if (cgs_iter % 10 == 0) {
                printf("Pressure Solver Iteration=%d; CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e\n", pressureIterations + 1, cgs_iter, r_norm / r_norm_init, r_norm);
            }

            if ((r_norm / r_norm_init > SolverConfig.solverToleranceRel) && (r_norm > SolverConfig.solverToleranceAbs)) {
                //Only continues if not yet within tolerance
                divide << <1, 1 >> > (d_r_norm, d_r_norm_old, d_beta);//beta = (rk+1 dot rk+1) / (rk dot rk)
                hipDeviceSynchronize();
                scalarVectorMult_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_beta, d_pk, d_temp, d_SolverConfig); //temp=beta*pk
                hipDeviceSynchronize();
                addVectors_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_temp, d_rkp1, d_pk, d_SolverConfig); //pk+1=rk+1 + beta*pk 
                hipDeviceSynchronize();
            }
            else {
                printf("Pressure Solver Iteration=%d; CG Iteration=%d; RelRes=%0.2e;  AbsRes=%0.2e [Converged]\n", pressureIterations + 1, cgs_iter, r_norm / r_norm_init, r_norm);
                break;
            }
        }

        if (pressureIterations > 0) {
            //For iterations 2 and onwards, attempts to use over-relaxation to make convergence faster
            subtractVectors_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_PressureField, d_PressureFieldOld, d_temp, d_SolverConfig); //temp=P-Pold
            hipDeviceSynchronize();

            //Also computes the norm of the difference to assess convergence
            vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_temp, d_temp, d_temp_scal, d_SolverConfig); //temp_scal = (P-Pold) dot (P-Pold)
            hipMemcpy(&delta_p_norm, d_temp_scal, sizeof(varfloat), hipMemcpyDeviceToHost);
            delta_p_norm = sqrt(delta_p_norm);

            //Also gets the norm of the pressure field to assess convergence
            vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_PressureField, d_PressureField, d_temp_scal, d_SolverConfig); //temp_scal = P dot P
            hipMemcpy(&p_norm, d_temp_scal, sizeof(varfloat), hipMemcpyDeviceToHost);
            p_norm = sqrt(p_norm);

            scalarVectorMult_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_over_relax, d_temp, d_temp, d_SolverConfig); //temp=over-relax * temp
            hipDeviceSynchronize();
            addVectors_GPU << <numBlocks1D, threadsPerBlock1D >> > (d_PressureFieldOld, d_temp, d_PressureField, d_SolverConfig); //P=Pold+overRelax*temp
            hipDeviceSynchronize();
        }

        hipDeviceSynchronize();
        hipMemcpyAsync(d_PressureFieldOld, d_PressureField, boxArraySize, hipMemcpyDeviceToDevice, stream[0]); //Copies pressure field to pressure_old
        vectorDot_GPU << <numBlocks1D, threadsPerBlock1D, 0, stream[1] >> > (d_PressureField, d_PressureField, d_temp_scal, d_SolverConfig); //Takes the norm of the Pressure Field x_k for output
        hipDeviceSynchronize();
        varfloat x_norm; hipMemcpy(&x_norm, d_temp_scal, sizeof(varfloat), hipMemcpyDeviceToHost);
        x_norm = sqrt(x_norm);
        varfloat relativeResidualNorm = abs(r_norm_init / first_r_norm_init);
        printf("Relative pressure residual norm:%f)\n", relativeResidualNorm);

        varfloat relativePressureNorm = abs(delta_p_norm / p_norm);
        printf("Relative pressure difference norm:%f)\n", relativePressureNorm);

        //checks if change is less than tolerance
        if (relativeResidualNorm < SolverConfig.PressureSolverToleranceRel) {
            printf("Pressure field norm change below tolerance. Exiting.\n");
            break;
        }

        //Provides user with timing for this round
        ClockToc("CG Solver Iteration Time:");


        //Prints out info about iterations to csv file
        toc2 = clock() - tic2;
        timeTask = ((varfloat)toc2) / CLOCKS_PER_SEC; // in seconds
        string stringOut = ""; stringOut.append(to_string(pressureIterations + 1)); stringOut.append(",");
        stringOut.append(to_string(relativeResidualNorm)); stringOut.append(",");
        stringOut.append(to_string(relativePressureNorm)); stringOut.append(",");
        stringOut.append(to_string(timeTask)); stringOut.append("\n");
        csvOutput->write(stringOut.c_str(), stringOut.size() * sizeof(char));

        //Saves a checkpoint iteration
        if ((SolverConfig.CheckpointIterations > 0) && ((pressureIterations + 1) % SolverConfig.CheckpointIterations == 0)) {
            printf("\n~~~Saving checkpoint iteration %d...~~\n", pressureIterations + 1);
            hipMemcpy(PressureField, d_PressureField, boxArraySize, hipMemcpyDeviceToHost);
            SaveVTK(OutputFileName, PressureField, SolverConfig, VTK_Contents);
            printf("\nSaved, Successfully!\n");
        }
    }

    //Extracts 3D array from GPU Memory
    hipMemcpy(PressureField, d_PressureField, boxArraySize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //Frees memory 
    printf("==========================================================\n");
    hipFree(d_PressureField); hipFree(d_RHS); hipFree(d_rk); hipFree(d_rkp1); hipFree(d_pk); hipFree(d_temp);
    hipFree(d_Wxp); hipFree(d_Wxm); hipFree(d_Wyp); hipFree(d_Wym); hipFree(d_Wzp); hipFree(d_Wzm);
    hipFree(d_SourceX); hipFree(d_SourceY); hipFree(d_SourceZ);
    hipFree(d_gridCoeffs); hipFree(d_SolverConfig);
    hipFree(d_beta); hipFree(d_alpha); hipFree(d_r_norm); hipFree(d_r_norm_old); hipFree(d_temp_scal);

    //Destroy parallel streams
    for (int i = 0; i < nStreams; ++i) {
        hipStreamDestroy(stream[i]);
    }
}



#pragma endregion



int main() {
    //Assumes single precision
    SolverParameters<varfloat> SolverConfig;
    //Reads input file to configure cameras
    printf("==============Reading Input Parameter File...==============\n");
    string fileName = "Arguments.conf";
    if (!ParseInputParameterFile(fileName, SolverConfig)) {
        //Aborts; file format is wrong.
        printf("Error reading input file! Aborting.\n");
        return 0;
    }
    
    if (!ParseInputParameterFile(fileName, SolverConfig)) {
        //Aborts; file format is wrong.
        printf("Error reading input file! Aborting.\n");
        return 0;
    }


    //Figures out how many VTK files to render by looking into the filename convention
    vector <string> BoxDataFileList;
    vector <int> BoxDataFileNumber;
    path pathObj(SolverConfig.BoxInputFile);
    string BoxPath = pathObj.parent_path().string();
    string BoxFileName = pathObj.filename().string();

    //First let's see if the string <frame> is even used here
    size_t pos = BoxFileName.find("<frame>");
    if (pos == std::string::npos) {
        //String <frame> not used, so must be a single file.
        printf("No pattern <frame> found, so using the full file name below as the VTK file:\n %s\n", SolverConfig.BoxInputFile.c_str());
        BoxDataFileList.push_back(SolverConfig.BoxInputFile);
        BoxDataFileNumber.push_back(0);
    }
    else {
        //String <frame> not used, so let's find all instances of files using that string
        string BeforeFrame = BoxFileName.substr(0, pos);
        string AfterFrame = BoxFileName.substr(pos + 7, BoxFileName.length() - (pos + 7));

        replace(BoxFileName, "<frame>", "\\d+\\");
        regex regex_pattern(BoxFileName);

        printf("Searching Directory %s \n for files of the pattern %s...\n", BoxPath.c_str(), BoxFileName.c_str());
        for (const auto& entry : directory_iterator(BoxPath)) { //Goes through the directory and finds the files
            string ThisFileName = entry.path().filename().string();
            printf("Looking at file %s...\n", ThisFileName.c_str());
            if (is_regular_file(entry) && regex_match(ThisFileName, regex_pattern)) {
                string ss = ThisFileName;
                eraseSubStr(ss, BeforeFrame);
                eraseSubStr(ss, AfterFrame);

                int number = std::stoi(ss);
                printf("-- File matches expression. File Number = %d.\n", number);
                BoxDataFileList.push_back(entry.path().string());
                BoxDataFileNumber.push_back(number);
            }
        }
    }


    for (int vtkIdx = 0; vtkIdx < BoxDataFileList.size(); vtkIdx++) {
        int ThisFileNumber = BoxDataFileNumber[vtkIdx];
        string ThisBoxFile = BoxDataFileList[vtkIdx];

        printf("\n\n\n ========Initializing Solver for file '%s'========\n", ThisBoxFile.c_str());

        ClockTic();
        BoxContents<varfloat> VTK_Contents;
        //===========Reads the box contents file===========
        if (!ReadVTK(ThisBoxFile, VTK_Contents)) {
            //Aborts; file format is wrong.
            printf("Error reading box data file! Aborting.\n");
            return 0;
        }
        SolverConfig.BoxGridPoints = VTK_Contents.BoxGridSize; // Ensure the variables have the same value
        SolverConfig.GridDelta = VTK_Contents.GridDelta;
        SolverConfig.totalBoxElements = VTK_Contents.totalBoxElements;

        //Configure output file for VolumeSum metric results
        ofstream csvOutput("Residual_RelativeNorm.csv", std::ios::out);
        if (!csvOutput) {
            printf("Error writing output CSV file!\n");
            return false;
        }
        string stringOut = "Iteration, Residual Norm, Pressure Difference Norm, Time [s]\n"; //header
        csvOutput.write(stringOut.c_str(), stringOut.size() * sizeof(char));

        ClockToc("File Read Time:");

        //===========Computes the weight coefficients===========
        printf("========Initializing Grid Weight Coefficients========\n");
        ClockTic();

        gridCoefficients<varfloat> gridCoeffs;
        if (VTK_Contents.BoxGridSize.z == 1) {
            //Coefficients are 2D
            gridCoefficients2D(VTK_Contents, gridCoeffs);
        }
        else {
            //Coefficients are 3D
            gridCoefficients3D(VTK_Contents, gridCoeffs, 1e7);
        }

        ClockToc("Grid Coefficient Computation Time:");

        //~~~~~~~~~~~~===========LOADS DATA INTO GPU AND FIRES UP THE CONJUGATE GRADIENT SOLVER===========~~~~~~~~~~~~~~~~~~~~~
        printf("========Starting up the solver...========\n");

        ClockTic();
        // Allocate CPU memory for the result
        varfloat* PressureField;
        long long boxArraySize = sizeof(varfloat) * VTK_Contents.totalBoxElements;
        PressureField = (varfloat*)malloc(boxArraySize);
        FillBox(PressureField, ZEROS, SolverConfig); // Initializes with zeros

        //Starts solving the pressure equations on a loop
        varfloat* RHS; RHS = (varfloat*)malloc(boxArraySize);
        FillBox(RHS, ZEROS, SolverConfig); // Initializes with zeros
        varfloat CurrentResultNorm = 0.0;

        // Precompute the weights
        BoxWeights< varfloat> Weights;
        Weights.Wxp = (varfloat*)malloc(boxArraySize); Weights.Wxm = (varfloat*)malloc(boxArraySize);
        Weights.Wyp = (varfloat*)malloc(boxArraySize); Weights.Wym = (varfloat*)malloc(boxArraySize);
        Weights.Wzp = (varfloat*)malloc(boxArraySize); Weights.Wzm = (varfloat*)malloc(boxArraySize);
        Precompute_Weights_Iteration_nPlusOne(Weights, SolverConfig, VTK_Contents, gridCoeffs);

        ClockToc("Weight Precomputation Time:");

        string OutputFileName = SolverConfig.BoxOutputFile;
        replace(OutputFileName, "<frame>", to_string(ThisFileNumber));

        ClockTic2();
        //Solves the equations with the solver
        ConjugateGradientSolver_GPU(PressureField, RHS, Weights, SolverConfig, VTK_Contents, gridCoeffs, OutputFileName, &csvOutput);
        ClockToc2("Total Time for Pressure Solver:");

        //===========Saves the result for the current file===========
        SaveVTK(OutputFileName, PressureField, SolverConfig, VTK_Contents);
        printf("\nSaved Output File Successfully!\n");

        free(PressureField); free(RHS);

        //Closes CSV output
        csvOutput.close();
    }

    return 0;
}

